#include "hip/hip_runtime.h"
/*
********************************************************************************
* Curve grid metric calculation using MacCormack scheme                        *
********************************************************************************
*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "netcdf.h"

#include "fdlib_mem.h"
#include "fdlib_math.h"
#include "fd_t.h"
#include "gd_t.h"
#include "constants.h"

int 
gd_curv_init(gdcurv_t *gdcurv)
{
  /*
   * 0-2: x3d, y3d, z3d
   */
  gdcurv->ncmp = CONST_NDIM;
  
  // vars
  gdcurv->v4d = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp * gdcurv->ncmp, 0.0, "gd_curv_init");
  if (gdcurv->v4d == NULL) {
      fprintf(stderr,"Error: failed to alloc coord vars\n");
      fflush(stderr);
  }
  
  // position of each v4d
  size_t *cmp_pos = (size_t *) fdlib_mem_calloc_1d_sizet(gdcurv->ncmp,
                                                         0,
                                                         "gd_curv_init");
  
  // name of each v4d
  char **cmp_name = (char **) fdlib_mem_malloc_2l_char(gdcurv->ncmp,
                                                       CONST_MAX_STRLEN,
                                                       "gd_curv_init");
  
  // set value
  int icmp = 0;
  cmp_pos[icmp] = icmp * gdcurv->siz_icmp;
  sprintf(cmp_name[icmp],"%s","x");
  gdcurv->x3d = gdcurv->v4d + cmp_pos[icmp];

  icmp += 1;
  cmp_pos[icmp] = icmp * gdcurv->siz_icmp;
  sprintf(cmp_name[icmp],"%s","y");
  gdcurv->y3d = gdcurv->v4d + cmp_pos[icmp];

  icmp += 1;
  cmp_pos[icmp] = icmp * gdcurv->siz_icmp;
  sprintf(cmp_name[icmp],"%s","z");
  gdcurv->z3d = gdcurv->v4d + cmp_pos[icmp];
  
  // set pointer
  gdcurv->cmp_pos  = cmp_pos;
  gdcurv->cmp_name = cmp_name;

  // alloc AABB vars
  gdcurv->cell_xmin = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp, 0.0, "gd_curv_init");
  gdcurv->cell_xmax = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp, 0.0, "gd_curv_init");
  gdcurv->cell_ymin = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp, 0.0, "gd_curv_init");
  gdcurv->cell_ymax = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp, 0.0, "gd_curv_init");
  gdcurv->cell_zmin = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp, 0.0, "gd_curv_init");
  gdcurv->cell_zmax = (float *) fdlib_mem_calloc_1d_float(
                  gdcurv->siz_icmp, 0.0, "gd_curv_init");
  if (gdcurv->cell_zmax == NULL) {
      fprintf(stderr,"Error: failed to alloc coord AABB vars\n");
      fflush(stderr);
  }

  gdcurv->tile_istart = (int *) fdlib_mem_calloc_1d_int(
                        GD_TILE_NX, 0.0, "gd_curv_init");
  gdcurv->tile_iend   = (int *) fdlib_mem_calloc_1d_int(
                        GD_TILE_NX, 0.0, "gd_curv_init");
  gdcurv->tile_jstart = (int *) fdlib_mem_calloc_1d_int(
                        GD_TILE_NY, 0.0, "gd_curv_init");
  gdcurv->tile_jend   = (int *) fdlib_mem_calloc_1d_int(
                        GD_TILE_NY, 0.0, "gd_curv_init");
  gdcurv->tile_kstart = (int *) fdlib_mem_calloc_1d_int(
                        GD_TILE_NZ, 0.0, "gd_curv_init");
  gdcurv->tile_kend   = (int *) fdlib_mem_calloc_1d_int(
                        GD_TILE_NZ, 0.0, "gd_curv_init");

  int size = GD_TILE_NX * GD_TILE_NY * GD_TILE_NZ;
  gdcurv->tile_xmin = (float *) fdlib_mem_calloc_1d_float(
                        size, 0.0, "gd_curv_init");
  gdcurv->tile_xmax = (float *) fdlib_mem_calloc_1d_float(
                        size, 0.0, "gd_curv_init");
  gdcurv->tile_ymin = (float *) fdlib_mem_calloc_1d_float(
                        size, 0.0, "gd_curv_init");
  gdcurv->tile_ymax = (float *) fdlib_mem_calloc_1d_float(
                        size, 0.0, "gd_curv_init");
  gdcurv->tile_zmin = (float *) fdlib_mem_calloc_1d_float(
                        size, 0.0, "gd_curv_init");
  gdcurv->tile_zmax = (float *) fdlib_mem_calloc_1d_float(
                        size, 0.0, "gd_curv_init");

  return 0;
}

int 
gd_curv_metric_init(gdcurv_t        *gdcurv,
                    gd_metric_t *metric)
{
  const int num_grid_vars = 10;
  /*
   * 0: jac
   * 1-3: xi_x, xi_y, xi_z
   * 4-6: eta_x, eta_y, eta_z
   * 7-9: zeta_x, zeta_y, zeta_z
   */

  metric->nx   = gdcurv->nx;
  metric->ny   = gdcurv->ny;
  metric->nz   = gdcurv->nz;
  metric->ncmp = num_grid_vars;

  metric->siz_iy  = metric->nx;
  metric->siz_iz  = metric->nx * metric->ny;
  metric->siz_icmp = metric->nx * metric->ny * metric->nz;
  
  // vars
  metric->v4d = (float *) fdlib_mem_calloc_1d_float(
                  metric->siz_icmp * metric->ncmp, 0.0, "gd_curv_init_g4d");
  if (metric->v4d == NULL) {
      fprintf(stderr,"Error: failed to alloc metric vars\n");
      fflush(stderr);
  }
  
  // position of each v4d
  size_t *cmp_pos = (size_t *) fdlib_mem_calloc_1d_sizet(metric->ncmp,
                                                         0, 
                                                         "gd_curv_metric_init");
  
  // name of each v4d
  char **cmp_name = (char **) fdlib_mem_malloc_2l_char(metric->ncmp,
                                                       CONST_MAX_STRLEN,
                                                       "gd_curv_metric_init");
  
  // set value
  for (int icmp=0; icmp < metric->ncmp; icmp++)
  {
    cmp_pos[icmp] = icmp * metric->siz_icmp;
  }

  int icmp = 0;
  sprintf(cmp_name[icmp],"%s","jac");
  metric->jac = metric->v4d + cmp_pos[icmp];

  icmp += 1;
  sprintf(cmp_name[icmp],"%s","xi_x");
  metric->xi_x = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","xi_y");
  metric->xi_y = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","xi_z");
  metric->xi_z = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","eta_x");
  metric->eta_x = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","eta_y");
  metric->eta_y = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","eta_z");
  metric->eta_z = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","zeta_x");
  metric->zeta_x = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","zeta_y");
  metric->zeta_y = metric->v4d + cmp_pos[icmp];
  
  icmp += 1;
  sprintf(cmp_name[icmp],"%s","zeta_z");
  metric->zeta_z = metric->v4d + cmp_pos[icmp];
  
  // set pointer
  metric->cmp_pos  = cmp_pos;
  metric->cmp_name = cmp_name;

  return 0;
}

//
// need to change to use fdlib_math.c
//
int
gd_curv_metric_cal(gdcurv_t    *gdcurv,
                   gd_metric_t *metric)
{
  int ni1 = gdcurv->ni1;
  int ni2 = gdcurv->ni2;
  int nj1 = gdcurv->nj1;
  int nj2 = gdcurv->nj2;
  int nk1 = gdcurv->nk1;
  int nk2 = gdcurv->nk2;
  int nx  = gdcurv->nx;
  int ny  = gdcurv->ny;
  int nz  = gdcurv->nz;
  size_t siz_iy   = gdcurv->siz_iy;
  size_t siz_iz   = gdcurv->siz_iz;
  size_t siz_icmp = gdcurv->siz_icmp;

  // point to each var
  float *x3d  = gdcurv->x3d;
  float *y3d  = gdcurv->y3d;
  float *z3d  = gdcurv->z3d;
  float *jac3d= metric->jac;
  float *xi_x = metric->xi_x;
  float *xi_y = metric->xi_y;
  float *xi_z = metric->xi_z;
  float *et_x = metric->eta_x;
  float *et_y = metric->eta_y;
  float *et_z = metric->eta_z;
  float *zt_x = metric->zeta_x;
  float *zt_y = metric->zeta_y;
  float *zt_z = metric->zeta_z;
  float *x3d_ptr;
  float *y3d_ptr;
  float *z3d_ptr; 
  float x_xi, x_et, x_zt;
  float y_xi, y_et, y_zt;
  float z_xi, z_et, z_zt;
  float jac;
  float vec1[3], vec2[3], vec3[3], vecg[3];

  for (size_t k = nk1; k <= nk2; k++){
    for (size_t j = nj1; j <= nj2; j++) {
      for (size_t i = ni1; i <= ni2; i++)
      {
        size_t iptr = i + j * siz_iy + k * siz_iz;

        x_xi = 0.0; x_et = 0.0; x_zt = 0.0;
        y_xi = 0.0; y_et = 0.0; y_zt = 0.0;
        z_xi = 0.0; z_et = 0.0; z_zt = 0.0;

        x3d_ptr = x3d + iptr;
        y3d_ptr = y3d + iptr;
        z3d_ptr = z3d + iptr;
        M_FD_SHIFT_PTR_CENTER(x_xi, x3d_ptr, 1);
        M_FD_SHIFT_PTR_CENTER(y_xi, y3d_ptr, 1);
        M_FD_SHIFT_PTR_CENTER(z_xi, z3d_ptr, 1);

        M_FD_SHIFT_PTR_CENTER(x_et, x3d_ptr, siz_iy);
        M_FD_SHIFT_PTR_CENTER(y_et, y3d_ptr, siz_iy);
        M_FD_SHIFT_PTR_CENTER(z_et, z3d_ptr, siz_iy);

        M_FD_SHIFT_PTR_CENTER(x_zt, x3d_ptr, siz_iz);
        M_FD_SHIFT_PTR_CENTER(y_zt, y3d_ptr, siz_iz);
        M_FD_SHIFT_PTR_CENTER(z_zt, z3d_ptr, siz_iz);

        vec1[0] = x_xi; vec1[1] = y_xi; vec1[2] = z_xi;
        vec2[0] = x_et; vec2[1] = y_et; vec2[2] = z_et;
        vec3[0] = x_zt; vec3[1] = y_zt; vec3[2] = z_zt;

        fdlib_math_cross_product(vec1, vec2, vecg);
        jac = fdlib_math_dot_product(vecg, vec3);
        jac3d[iptr]  = jac;

        fdlib_math_cross_product(vec2, vec3, vecg);
        xi_x[iptr] = vecg[0] / jac;
        xi_y[iptr] = vecg[1] / jac;
        xi_z[iptr] = vecg[2] / jac;

        fdlib_math_cross_product(vec3, vec1, vecg);
        et_x[iptr] = vecg[0] / jac;
        et_y[iptr] = vecg[1] / jac;
        et_z[iptr] = vecg[2] / jac;

        fdlib_math_cross_product(vec1, vec2, vecg);
        zt_x[iptr] = vecg[0] / jac;
        zt_y[iptr] = vecg[1] / jac;
        zt_z[iptr] = vecg[2] / jac;
      }
    }
  }
    
  //mirror_symmetry(gdcurv,metric->v4d,metric->ncmp);
  geometric_symmetry(gdcurv,metric->v4d,metric->ncmp);

  return 0;
}

//
// exchange metics/coords
//
int
gd_curv_exchange(gdcurv_t *gdcurv,
                 float *g3d,
                 int ncmp,
                 int *neighid,
                 MPI_Comm topocomm)
{
  int nx  = gdcurv->nx;
  int ny  = gdcurv->ny;
  int nz  = gdcurv->nz;
  int ni1 = gdcurv->ni1;
  int ni2 = gdcurv->ni2;
  int nj1 = gdcurv->nj1;
  int nj2 = gdcurv->nj2;
  int nk1 = gdcurv->nk1;
  int nk2 = gdcurv->nk2;

  size_t siz_iy   = gdcurv->siz_iy;
  size_t siz_iz   = gdcurv->siz_iz;
  size_t siz_icmp = gdcurv->siz_icmp;

  // extend to ghosts, using mpi exchange
  // NOTE in different myid, nx(or ny) may not equal
  // so send type DTypeXL not equal recv type DTypeXL
  size_t s_iptr;
  size_t r_iptr;

  MPI_Status status;
  MPI_Datatype DTypeXL, DTypeYL, DTypeZL;

  MPI_Type_vector(ny*nz,
                  3,
                  nx,
                  MPI_FLOAT,
                  &DTypeXL);
  MPI_Type_vector(nz,
                  3*nx,
                  nx*ny,
                  MPI_FLOAT,
                  &DTypeYL);
  MPI_Type_vector(3,
                  nx*ny,
                  nx*ny,
                  MPI_FLOAT,
                  &DTypeZL);
  MPI_Type_commit(&DTypeXL);
  MPI_Type_commit(&DTypeYL);
  MPI_Type_commit(&DTypeZL);

  for(int i=0; i<ncmp; i++)
  {
    //bdry x1 to bdry x2
    s_iptr = ni1 + i * siz_icmp;        //sendbuff point (ni1,ny1,nz1)
    r_iptr = (ni2+1) + i * siz_icmp;    //recvbuff point (ni2+1,ny1,nz1)
    MPI_Sendrecv(&g3d[s_iptr],1,DTypeXL,neighid[0],110,
                 &g3d[r_iptr],1,DTypeXL,neighid[1],110,
                 topocomm,&status);
    //bdry x2 to bdry x1
    s_iptr = (ni2-3+1) + i * siz_icmp;    //sendbuff point (ni2-3+1,ny1,nz1)
    r_iptr = (ni1-3) + i * siz_icmp;      //recvbuff point (ni1-3,ny1,nz1)
    MPI_Sendrecv(&g3d[s_iptr],1,DTypeXL,neighid[1],120,
                 &g3d[r_iptr],1,DTypeXL,neighid[0],120,
                 topocomm,&status);
    //bdry y1 to bdry y2
    s_iptr = nj1 * siz_iy + i * siz_icmp;        //sendbuff point (nx1,nj1,nz1)
    r_iptr = (nj2+1) * siz_iy + i * siz_icmp;    //recvbuff point (nx1,nj2+1,nz1)
    MPI_Sendrecv(&g3d[s_iptr],1,DTypeYL,neighid[2],210,
                 &g3d[r_iptr],1,DTypeYL,neighid[3],210,
                 topocomm,&status);
    //bdry y2 to bdry y1
    s_iptr = (nj2-3+1) * siz_iy + i * siz_icmp;   //sendbuff point (nx1,nj2-3+1,nz1)
    r_iptr = (nj1-3) * siz_iy + i * siz_icmp;     //recvbuff point (nx1,nj1-3,nz1)
    MPI_Sendrecv(&g3d[s_iptr],1,DTypeYL,neighid[3],220,
                 &g3d[r_iptr],1,DTypeYL,neighid[2],220,
                 topocomm,&status);
    //bdry z1 to bdry z2
    s_iptr = nk1 * siz_iz + i * siz_icmp;        //sendbuff point (nx1,ny1,nk1)
    r_iptr = (nk2+1) * siz_iz + i * siz_icmp;    //recvbuff point (nx1,ny1,nk2+1)
    MPI_Sendrecv(&g3d[s_iptr],1,DTypeZL,neighid[4],310,
                 &g3d[r_iptr],1,DTypeZL,neighid[5],310,
                 topocomm,&status);
    //bdry z2 to bdry z1
    s_iptr = (nk2-3+1) * siz_iz + i * siz_icmp;   //sendbuff point (nx1,ny1,nk2-3+1)
    r_iptr = (nk1-3) * siz_iz + i * siz_icmp;     //recvbuff point (nx1,ny1,nk1-3)
    MPI_Sendrecv(&g3d[s_iptr],1,DTypeZL,neighid[5],320,
                 &g3d[r_iptr],1,DTypeZL,neighid[4],320,
                 topocomm,&status);
  }

  return 0;
}

int
gd_curv_gen_fault(gdcurv_t *gdcurv,
                  int fault_i_global_indx,
                  float dh,
                  char *in_grid_fault_nc)
{
  int nx = gdcurv->nx;
  int ny = gdcurv->ny;
  int nz = gdcurv->nz;

  int ni = gdcurv->ni;
  int nj = gdcurv->nj;
  int nk = gdcurv->nk;

  int ni1 = gdcurv->ni1;
  int ni2 = gdcurv->ni2;
  int nj1 = gdcurv->nj1;
  int nj2 = gdcurv->nj2;
  int nk1 = gdcurv->nk1;
  int nk2 = gdcurv->nk2;
  int gni1 = gdcurv->gni1;
  int npoint_x = gdcurv->npoint_x;

  size_t siz_iy  = gdcurv->siz_iy;
  size_t siz_iz  = gdcurv->siz_iz;
  size_t iptr, iptr_b, iptr_c;

  float *x3d = gdcurv->x3d;
  float *y3d = gdcurv->y3d;
  float *z3d = gdcurv->z3d;

  float *fault_x = (float *) malloc(sizeof(float)*nj*nk);
  float *fault_y = (float *) malloc(sizeof(float)*nj*nk);
  float *fault_z = (float *) malloc(sizeof(float)*nj*nk);
  float *xline   = (float *) malloc(sizeof(float)*nx);

  nc_read_fault_geometry(fault_x, fault_y, fault_z, in_grid_fault_nc, gdcurv);

  int i0 = fault_i_global_indx + 3;  // now with ghost index 
  xline[i0] = 0.0;
  int width1 = 10;
  int width2 = 55;
  float compr;
  int dist;
  for(int i = i0-1; i>=ni1; i--)
  {
    dist = abs(i-i0); 
    if(dist < width1)
    {
      compr = 0;
    }
    if(dist>=width1 && dist < width2)
    {
      compr = 1.0 - cos(PI * (i - (i0 - width1))/(float)(width2-width1));
    }
    if(dist >= width2)
    {
      compr = 2.0;
    }
    compr = 0.5 + 0.25 * compr;
    xline[i] = xline[i+1] - dh *compr;
  }
  for(int i = i0+1; i<=ni2; i++)
  {
    dist = abs(i-i0); 
    if(dist < width1)
    {
      compr = 0;
    }
    if(dist>=width1 && dist < width2)
    {
      compr = 1.0 - cos(PI * (i - (i0 + width1))/(float)(width2-width1));
    }
    if(dist >= width2)
    {
      compr = 2.0;
    }
    compr = 0.5 + 0.25 * compr;
    xline[i] = xline[i-1] + dh *compr;
  }

  for (int k = nk1; k <= nk2; k++){
    for (int j = nj1; j <= nj2; j++){
      for (int i = ni1; i <= ni2; i++){

        //int gi = gni1 + i - 3; 
        //float x = fault_x[j-3 + (k-3) * nj] + gi * dh + x0;

        float x = fault_x[j-3 + (k-3) * nj] + xline[i];
        float y = fault_y[j-3 + (k-3) * nj];
        float z = fault_z[j-3 + (k-3) * nj];

        iptr = i + j * siz_iy + k * siz_iz;
        x3d[iptr] = x;
        y3d[iptr] = y;
        z3d[iptr] = z;
      }
    }
  }

  // extend to ghosts. 
  geometric_symmetry(gdcurv,gdcurv->v4d,gdcurv->ncmp);
   
  free(fault_x);
  free(fault_y);
  free(fault_z);
  free(xline);

  return 0;
}

int
nc_read_fault_geometry(float *fault_x, float *fault_y, float *fault_z,
                       char *in_grid_fault_nc, gdcurv_t *gdcurv)
{
  int nj = gdcurv->nj;
  int nk = gdcurv->nk;

  // thisid dimension 0, 1, 2, thisid[2] vary first
  int ierr;
  int ncid;
  int xid, yid, zid;
  size_t start[] = {gdcurv->gnk1, gdcurv->gnj1};
  size_t count[] = {nk, nj};
  
  ierr = nc_open(in_grid_fault_nc, NC_NOWRITE, &ncid); handle_nc_err(ierr);

  //read coords
  ierr = nc_inq_varid(ncid, "x", &xid); handle_nc_err(ierr);
  ierr = nc_inq_varid(ncid, "y", &yid); handle_nc_err(ierr);
  ierr = nc_inq_varid(ncid, "z", &zid); handle_nc_err(ierr);

  ierr = nc_get_vara_float(ncid, xid, start, count, fault_x); handle_nc_err(ierr);
  ierr = nc_get_vara_float(ncid, yid, start, count, fault_y); handle_nc_err(ierr);
  ierr = nc_get_vara_float(ncid, zid, start, count, fault_z); handle_nc_err(ierr);

  ierr = nc_close(ncid); handle_nc_err(ierr);

  return 0;
}

int
mirror_symmetry(gdcurv_t *gdcurv, float *v4d, int ncmp)
{
  int ni1 = gdcurv->ni1;
  int ni2 = gdcurv->ni2;
  int nj1 = gdcurv->nj1;
  int nj2 = gdcurv->nj2;
  int nk1 = gdcurv->nk1;
  int nk2 = gdcurv->nk2;
  int nx  = gdcurv->nx;
  int ny  = gdcurv->ny;
  int nz  = gdcurv->nz;
  size_t siz_iy  = gdcurv->siz_iy;
  size_t siz_iz  = gdcurv->siz_iz;
  size_t siz_icmp  = gdcurv->siz_icmp;

  size_t iptr, iptr1, iptr2; 
  for(int icmp=0; icmp<ncmp; icmp++){
    iptr = icmp * siz_icmp;
    // x1, mirror
    for (size_t k = 0; k < nz; k++){
      for (size_t j = 0; j < ny; j++){
        for (size_t i = 0; i < ni1; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + (2*ni1-i) + j * siz_iy +  k * siz_iz;
          v4d[iptr1] = v4d[iptr2];
        }
      }
    }
    // x2, mirror
    for (size_t k = 0; k < nz; k++){
      for (size_t j = 0; j < ny; j++){
        for (size_t i = ni2+1; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + (2*ni2-i) + j * siz_iy + k * siz_iz;
          v4d[iptr1] = v4d[iptr2];
        }
      }
    }
    // y1, mirror
    for (size_t k = 0; k < nz; k++){
      for (size_t j = 0; j < nj1; j++){
        for (size_t i = 0; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + (2*nj1-j) * siz_iy +  k * siz_iz;
          v4d[iptr1] = v4d[iptr2];
        }
      }
    }
    // y2, mirror
    for (size_t k = 0; k < nz; k++){
      for (size_t j = nj2+1; j < ny; j++){
        for (size_t i = 0; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + (2*nj2-j) * siz_iy +  k * siz_iz;
          v4d[iptr1] = v4d[iptr2];
        }
      }
    }
    // z1, mirror
    for (size_t k = 0; k < nk1; k++) {
      for (size_t j = 0; j < ny; j++) {
        for (size_t i = 0; i < nx; i++) {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + j * siz_iy + (2*nk1-k) * siz_iz;
          v4d[iptr1] = v4d[iptr2];
        }
      }
    }
    // z2, mirror
    for (size_t k = nk2+1; k < nz; k++) {
      for (size_t j = 0; j < ny; j++) {
        for (size_t i = 0; i < nx; i++) {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + j * siz_iy + (2*nk2-k) * siz_iz;
          v4d[iptr1] = v4d[iptr2];
        }
      }
    }
  }

  return 0;
}

int 
geometric_symmetry(gdcurv_t *gdcurv,float *v4d, int ncmp)
{
  int ni1 = gdcurv->ni1;
  int ni2 = gdcurv->ni2;
  int nj1 = gdcurv->nj1;
  int nj2 = gdcurv->nj2;
  int nk1 = gdcurv->nk1;
  int nk2 = gdcurv->nk2;
  int nx  = gdcurv->nx;
  int ny  = gdcurv->ny;
  int nz  = gdcurv->nz;
  size_t siz_iy  = gdcurv->siz_iy;
  size_t siz_iz  = gdcurv->siz_iz;
  size_t siz_icmp  = gdcurv->siz_icmp;

  size_t iptr, iptr1, iptr2, iptr3; 
  for(int icmp=0; icmp<ncmp; icmp++){
    iptr = icmp * siz_icmp;
    // x1 
    for (size_t k = 0; k < nz; k++){
      for (size_t j = 0; j < ny; j++){
        for (size_t i = 0; i < ni1; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + ni1 + j * siz_iy + k * siz_iz;
          iptr3 = iptr + (2*ni1-i) + j * siz_iy + k * siz_iz;
          v4d[iptr1] = 2*v4d[iptr2] - v4d[iptr3];
        }
      }
    }
    // x2
    for (size_t k = 0; k < nz; k++){
      for (size_t j = 0; j < ny; j++){
        for (size_t i = ni2+1; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + ni2 + j * siz_iy + k * siz_iz;
          iptr3 = iptr + (2*ni2-i) + j * siz_iy + k * siz_iz;
          v4d[iptr1] = 2*v4d[iptr2] - v4d[iptr3];
        }
      }
    }
    // y1 
    for (size_t k = 0; k < nz; k++){
      for (size_t j = 0; j < nj1; j++){
        for (size_t i = 0; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + nj1 * siz_iy + k * siz_iz;
          iptr3 = iptr + i + (2*nj1-j) * siz_iy + k * siz_iz;
          v4d[iptr1] = 2*v4d[iptr2] - v4d[iptr3];
        }
      }
    }
    // y2 
    for (size_t k = 0; k < nz; k++){
      for (size_t j = nj2+1; j < ny; j++){
        for (size_t i = 0; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + nj2 * siz_iy + k * siz_iz;
          iptr3 = iptr + i + (2*nj2-j) * siz_iy + k * siz_iz;
          v4d[iptr1] = 2*v4d[iptr2] - v4d[iptr3];
        }
      }
    }
    // z1
    for (size_t k = 0; k < nk1; k++){
      for (size_t j = 0; j < ny; j++){
        for (size_t i = 0; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + j * siz_iy + nk1 * siz_iz;
          iptr3 = iptr + i + j * siz_iy + (2*nk1-k) * siz_iz;
          v4d[iptr1] = 2*v4d[iptr2] - v4d[iptr3];
        }
      }
    }
    // z2
    for (size_t k = nk2+1; k < nz; k++) {
      for (size_t j = 0; j < ny; j++){
        for (size_t i = 0; i < nx; i++)
        {
          iptr1 = iptr + i + j * siz_iy + k * siz_iz;
          iptr2 = iptr + i + j * siz_iy + nk2 * siz_iz;
          iptr3 = iptr + i + j * siz_iy + (2*nk2-k) * siz_iz;
          v4d[iptr1] = 2*v4d[iptr2] - v4d[iptr3];
        }
      }
    }
  }

  return 0;
}

//
// input/output
//
int
gd_curv_coord_export(gdcurv_t *gdcurv,
                     char *fname_coords,
                     char *output_dir)
{
  size_t *c3d_pos   = gdcurv->cmp_pos;
  char  **c3d_name  = gdcurv->cmp_name;
  int number_of_vars = gdcurv->ncmp;
  int nx = gdcurv->nx;
  int ny = gdcurv->ny;
  int nz = gdcurv->nz;
  int ni1 = gdcurv->ni1;
  int nj1 = gdcurv->nj1;
  int nk1 = gdcurv->nk1;
  int ni  = gdcurv->ni;
  int nj  = gdcurv->nj;
  int nk  = gdcurv->nk;
  int gni1 = gdcurv->ni1_to_glob_phys0;
  int gnj1 = gdcurv->nj1_to_glob_phys0;
  int gnk1 = gdcurv->nk1_to_glob_phys0;

  // construct file name
  char ou_file[CONST_MAX_STRLEN];
  sprintf(ou_file, "%s/coord_%s.nc", output_dir, fname_coords);
  
  // read in nc
  int ncid;
  int varid[gdcurv->ncmp];
  int dimid[CONST_NDIM];

  int ierr = nc_create(ou_file, NC_CLOBBER | NC_64BIT_OFFSET, &ncid); handle_nc_err(ierr);

  // define dimension
  ierr = nc_def_dim(ncid, "i", nx, &dimid[2]);
  ierr = nc_def_dim(ncid, "j", ny, &dimid[1]);
  ierr = nc_def_dim(ncid, "k", nz, &dimid[0]);

  // define vars
  for (int ivar=0; ivar<gdcurv->ncmp; ivar++) {
    ierr = nc_def_var(ncid, gdcurv->cmp_name[ivar], NC_FLOAT, CONST_NDIM, dimid, &varid[ivar]);
    handle_nc_err(ierr);
  }

  // attribute: index in output snapshot, index w ghost in thread
  int l_start[] = { ni1, nj1, nk1 };
  nc_put_att_int(ncid,NC_GLOBAL,"local_index_of_first_physical_points",
                   NC_INT,CONST_NDIM,l_start);

  int g_start[] = { gni1, gnj1, gnk1 };
  nc_put_att_int(ncid,NC_GLOBAL,"global_index_of_first_physical_points",
                   NC_INT,CONST_NDIM,g_start);

  int l_count[] = { ni, nj, nk };
  nc_put_att_int(ncid,NC_GLOBAL,"count_of_physical_points",
                   NC_INT,CONST_NDIM,l_count);

  // end def
  ierr = nc_enddef(ncid);  handle_nc_err(ierr);

  // add vars
  for (int ivar=0; ivar<gdcurv->ncmp; ivar++) {
    float *ptr = gdcurv->v4d + gdcurv->cmp_pos[ivar];
    ierr = nc_put_var_float(ncid, varid[ivar],ptr);
    handle_nc_err(ierr);
  }
  
  // close file
  ierr = nc_close(ncid); handle_nc_err(ierr);

  return 0;
}

int
gd_curv_coord_import(gdcurv_t *gdcurv, char *fname_coords, char *import_dir)
{
  // construct file name
  char in_file[CONST_MAX_STRLEN];
  sprintf(in_file, "%s/coord_%s.nc", import_dir, fname_coords);
  
  // read in nc
  int ncid;
  int varid;

  int ierr = nc_open(in_file, NC_NOWRITE, &ncid); handle_nc_err(ierr);

  // read vars
  for (int ivar=0; ivar<gdcurv->ncmp; ivar++)
  {
    float *ptr = gdcurv->v4d + gdcurv->cmp_pos[ivar];
    ierr = nc_inq_varid(ncid, gdcurv->cmp_name[ivar], &varid); 
    handle_nc_err(ierr);
    ierr = nc_get_var(ncid, varid, ptr); handle_nc_err(ierr);
  }
  
  // close file
  ierr = nc_close(ncid); handle_nc_err(ierr);

  return 0;
}


int
gd_curv_metric_export(gdcurv_t        *gdcurv,
                      gd_metric_t *metric,
                      char *fname_coords,
                      char *output_dir)
{
  size_t *g3d_pos   = metric->cmp_pos;
  char  **g3d_name  = metric->cmp_name;
  int  number_of_vars = metric->ncmp;
  int  nx = metric->nx;
  int  ny = metric->ny;
  int  nz = metric->nz;
  int  ni1 = gdcurv->ni1;
  int  nj1 = gdcurv->nj1;
  int  nk1 = gdcurv->nk1;
  int  ni  = gdcurv->ni;
  int  nj  = gdcurv->nj;
  int  nk  = gdcurv->nk;
  int  gni1 = gdcurv->ni1_to_glob_phys0;
  int  gnj1 = gdcurv->nj1_to_glob_phys0;
  int  gnk1 = gdcurv->nk1_to_glob_phys0;

  // construct file name
  char ou_file[CONST_MAX_STRLEN];
  sprintf(ou_file, "%s/metric_%s.nc", output_dir, fname_coords);
  
  // read in nc
  int ncid;
  int varid[number_of_vars];
  int dimid[CONST_NDIM];

  int ierr = nc_create(ou_file, NC_CLOBBER | NC_64BIT_OFFSET, &ncid); handle_nc_err(ierr);

  // define dimension
  ierr = nc_def_dim(ncid, "i", nx, &dimid[2]);
  ierr = nc_def_dim(ncid, "j", ny, &dimid[1]);
  ierr = nc_def_dim(ncid, "k", nz, &dimid[0]);

  // define vars
  for (int ivar=0; ivar<number_of_vars; ivar++) {
    ierr = nc_def_var(ncid, g3d_name[ivar], NC_FLOAT, CONST_NDIM, dimid, &varid[ivar]);
    handle_nc_err(ierr);
  }

  // attribute: index in output snapshot, index w ghost in thread
  int l_start[] = { ni1, nj1, nk1 };
  nc_put_att_int(ncid,NC_GLOBAL,"local_index_of_first_physical_points",
                   NC_INT,CONST_NDIM,l_start);

  int g_start[] = { gni1, gnj1, gnk1 };
  nc_put_att_int(ncid,NC_GLOBAL,"global_index_of_first_physical_points",
                   NC_INT,CONST_NDIM,g_start);

  int l_count[] = { ni, nj, nk };
  nc_put_att_int(ncid,NC_GLOBAL,"count_of_physical_points",
                   NC_INT,CONST_NDIM,l_count);

  // end def
  ierr = nc_enddef(ncid);  handle_nc_err(ierr);

  // add vars
  for (int ivar=0; ivar<number_of_vars; ivar++) {
    float *ptr = metric->v4d + g3d_pos[ivar];
    ierr = nc_put_var_float(ncid, varid[ivar],ptr);
    handle_nc_err(ierr);
  }
  
  // close file
  ierr = nc_close(ncid); handle_nc_err(ierr);

  return 0;
}

int
gd_curv_metric_import(gd_metric_t *metric, char *fname_coords, char *import_dir)
{
  // construct file name
  char in_file[CONST_MAX_STRLEN];
  sprintf(in_file, "%s/metric_%s.nc", import_dir, fname_coords);
  
  // read in nc
  int ncid;
  int varid;

  int ierr = nc_open(in_file, NC_NOWRITE, &ncid); handle_nc_err(ierr);

  // read vars
  for (int ivar=0; ivar<metric->ncmp; ivar++)
  {
    float *ptr = metric->v4d + metric->cmp_pos[ivar];

    ierr = nc_inq_varid(ncid, metric->cmp_name[ivar], &varid); handle_nc_err(ierr);

    ierr = nc_get_var(ncid, varid, ptr); handle_nc_err(ierr);
  }
  
  // close file
  ierr = nc_close(ncid); handle_nc_err(ierr);

  return 0;
}

/*
 * set min/max of grid for loc
 */
int
gd_curv_set_minmax(gdcurv_t *gdcurv)
{
  // all points including ghosts
  float xmin = gdcurv->x3d[0], xmax = gdcurv->x3d[0];
  float ymin = gdcurv->y3d[0], ymax = gdcurv->y3d[0];
  float zmin = gdcurv->z3d[0], zmax = gdcurv->z3d[0];
  for (size_t i = 0; i < gdcurv->siz_icmp; i++){
      xmin = xmin < gdcurv->x3d[i] ? xmin : gdcurv->x3d[i];
      xmax = xmax > gdcurv->x3d[i] ? xmax : gdcurv->x3d[i];
      ymin = ymin < gdcurv->y3d[i] ? ymin : gdcurv->y3d[i];
      ymax = ymax > gdcurv->y3d[i] ? ymax : gdcurv->y3d[i];
      zmin = zmin < gdcurv->z3d[i] ? zmin : gdcurv->z3d[i];
      zmax = zmax > gdcurv->z3d[i] ? zmax : gdcurv->z3d[i];
  }
  gdcurv->xmin = xmin;
  gdcurv->xmax = xmax;
  gdcurv->ymin = ymin;
  gdcurv->ymax = ymax;
  gdcurv->zmin = zmin;
  gdcurv->zmax = zmax;

  // all physics points without ghosts
  xmin = gdcurv->xmax;
  xmax = gdcurv->xmin;
  ymin = gdcurv->ymax;
  ymax = gdcurv->ymin;
  zmin = gdcurv->zmax;
  zmax = gdcurv->zmin;
  for (int k = gdcurv->nk1; k <= gdcurv->nk2; k++) {
    for (int j = gdcurv->nj1; j <= gdcurv->nj2; j++) {
      for (int i = gdcurv->ni1; i <= gdcurv->ni2; i++) {
         size_t iptr = i + j * gdcurv->siz_iy + k * gdcurv->siz_iz;
         xmin = xmin < gdcurv->x3d[iptr] ? xmin : gdcurv->x3d[iptr];
         xmax = xmax > gdcurv->x3d[iptr] ? xmax : gdcurv->x3d[iptr];
         ymin = ymin < gdcurv->y3d[iptr] ? ymin : gdcurv->y3d[iptr];
         ymax = ymax > gdcurv->y3d[iptr] ? ymax : gdcurv->y3d[iptr];
         zmin = zmin < gdcurv->z3d[iptr] ? zmin : gdcurv->z3d[iptr];
         zmax = zmax > gdcurv->z3d[iptr] ? zmax : gdcurv->z3d[iptr];
      }
    }
  }
  gdcurv->xmin_phy = xmin;
  gdcurv->xmax_phy = xmax;
  gdcurv->ymin_phy = ymin;
  gdcurv->ymax_phy = ymax;
  gdcurv->zmin_phy = zmin;
  gdcurv->zmax_phy = zmax;

  // set cell range, last cell along each dim unusage
  for (int k = 0; k < gdcurv->nz-1; k++) {
    for (int j = 0; j < gdcurv->ny-1; j++) {
      for (int i = 0; i < gdcurv->nx-1; i++) {
         size_t iptr = i + j * gdcurv->siz_iy + k * gdcurv->siz_iz;
         xmin = gdcurv->x3d[iptr];
         ymin = gdcurv->y3d[iptr];
         zmin = gdcurv->z3d[iptr];
         xmax = xmin;
         ymax = ymin;
         zmax = zmin;
         for (int n3=0; n3<2; n3++) {
         for (int n2=0; n2<2; n2++) {
         for (int n1=0; n1<2; n1++) {
           size_t iptr_pt = iptr + n3 * gdcurv->siz_iz + n2 * gdcurv->siz_iy + n1;
           xmin = xmin < gdcurv->x3d[iptr_pt] ? xmin : gdcurv->x3d[iptr_pt];
           xmax = xmax > gdcurv->x3d[iptr_pt] ? xmax : gdcurv->x3d[iptr_pt];
           ymin = ymin < gdcurv->y3d[iptr_pt] ? ymin : gdcurv->y3d[iptr_pt];
           ymax = ymax > gdcurv->y3d[iptr_pt] ? ymax : gdcurv->y3d[iptr_pt];
           zmin = zmin < gdcurv->z3d[iptr_pt] ? zmin : gdcurv->z3d[iptr_pt];
           zmax = zmax > gdcurv->z3d[iptr_pt] ? zmax : gdcurv->z3d[iptr_pt];
         }
         }
         }
         gdcurv->cell_xmin[iptr] = xmin;
         gdcurv->cell_xmax[iptr] = xmax;
         gdcurv->cell_ymin[iptr] = ymin;
         gdcurv->cell_ymax[iptr] = ymax;
         gdcurv->cell_zmin[iptr] = zmin;
         gdcurv->cell_zmax[iptr] = zmax;
      }
    }
  }

  // set tile range 

  // partition into average plus left at last
  int nx_avg  = gdcurv->ni / GD_TILE_NX; // only for physcial points
  int nx_left = gdcurv->ni % GD_TILE_NX;
  int ny_avg  = gdcurv->nj / GD_TILE_NY;
  int ny_left = gdcurv->nj % GD_TILE_NY;
  int nz_avg  = gdcurv->nk / GD_TILE_NZ;
  int nz_left = gdcurv->nk % GD_TILE_NZ;
  for (int k_tile = 0; k_tile < GD_TILE_NZ; k_tile++)
  {
    if (k_tile == 0) {
      gdcurv->tile_kstart[k_tile] = gdcurv->nk1;
    } else {
      gdcurv->tile_kstart[k_tile] = gdcurv->tile_kend[k_tile-1] + 1;
    }

    gdcurv->tile_kend  [k_tile] = gdcurv->tile_kstart[k_tile] + nz_avg -1;
    if (k_tile < nz_left) {
      gdcurv->tile_kend[k_tile] += 1;
    }

    for (int j_tile = 0; j_tile < GD_TILE_NY; j_tile++)
    {
      if (j_tile == 0) {
        gdcurv->tile_jstart[j_tile] = gdcurv->nj1;
      } else {
        gdcurv->tile_jstart[j_tile] = gdcurv->tile_jend[j_tile-1] + 1;
      }

      gdcurv->tile_jend  [j_tile] = gdcurv->tile_jstart[j_tile] + ny_avg -1;
      if (j_tile < ny_left) {
        gdcurv->tile_jend[j_tile] += 1;
      }

      for (int i_tile = 0; i_tile < GD_TILE_NX; i_tile++)
      {
        if (i_tile == 0) {
          gdcurv->tile_istart[i_tile] = gdcurv->ni1;
        } else {
          gdcurv->tile_istart[i_tile] = gdcurv->tile_iend[i_tile-1] + 1;
        }

        gdcurv->tile_iend  [i_tile] = gdcurv->tile_istart[i_tile] + nx_avg -1;
        if (i_tile < nx_left) {
          gdcurv->tile_iend[i_tile] += 1;
        }

        // use large value to init
        xmin = 1.0e26;
        ymin = 1.0e26;
        zmin = 1.0e26;
        xmax = -1.0e26;
        ymax = -1.0e26;
        zmax = -1.0e26;
        // for cells in each tile
        for (int k = gdcurv->tile_kstart[k_tile]; k <= gdcurv->tile_kend[k_tile]; k++)
        {
          size_t iptr_k = k * gdcurv->siz_iz;
          for (int j = gdcurv->tile_jstart[j_tile]; j <= gdcurv->tile_jend[j_tile]; j++)
          {
            size_t iptr_j = iptr_k + j * gdcurv->siz_iy;
            for (int i = gdcurv->tile_istart[i_tile]; i <= gdcurv->tile_iend[i_tile]; i++)
            {
              size_t iptr = i + iptr_j;
              xmin = xmin < gdcurv->cell_xmin[iptr] ? xmin : gdcurv->cell_xmin[iptr];
              xmax = xmax > gdcurv->cell_xmax[iptr] ? xmax : gdcurv->cell_xmax[iptr];
              ymin = ymin < gdcurv->cell_ymin[iptr] ? ymin : gdcurv->cell_ymin[iptr];
              ymax = ymax > gdcurv->cell_ymax[iptr] ? ymax : gdcurv->cell_ymax[iptr];
              zmin = zmin < gdcurv->cell_zmin[iptr] ? zmin : gdcurv->cell_zmin[iptr];
              zmax = zmax > gdcurv->cell_zmax[iptr] ? zmax : gdcurv->cell_zmax[iptr];
            }
          }
        }
        int iptr_tile = i_tile + j_tile * GD_TILE_NX + k_tile * GD_TILE_NX *GD_TILE_NY;
        gdcurv->tile_xmin[iptr_tile] = xmin;
        gdcurv->tile_xmax[iptr_tile] = xmax;
        gdcurv->tile_ymin[iptr_tile] = ymin;
        gdcurv->tile_ymax[iptr_tile] = ymax;
        gdcurv->tile_zmin[iptr_tile] = zmin;
        gdcurv->tile_zmax[iptr_tile] = zmax;

      }
    }
  } // k_tile

  return 0;
}

/*
 * convert curv coord to global index using MPI
 */

int
gd_curv_coord_to_glob_indx(gdcurv_t *gdcurv,
                           float sx,
                           float sy,
                           float sz,
                           MPI_Comm comm,
                           int myid,
                           int   *ou_si, int *ou_sj, int *ou_sk,
                           float *ou_sx_inc, float *ou_sy_inc, float *ou_sz_inc)
{
  int is_here = 0;
  
  //NOTE si_glob sj_glob sk_glob must less -3. due to ghost points length is 3.
  int si_glob = -1000;
  int sj_glob = -1000;
  int sk_glob = -1000;
  float sx_inc = 0.0;
  float sy_inc = 0.0;
  float sz_inc = 0.0;
  int si = 0;
  int sj = 0;
  int sk = 0;

  // if located in this thread
  is_here = gd_curv_coord_to_local_indx(gdcurv,sx,sy,sz,
                                    &si, &sj, &sk, &sx_inc, &sy_inc, &sz_inc);

  // if in this thread
  if ( is_here == 1)
  {
    // conver to global index
    si_glob = gd_info_ind_lcext2glphy_i(si, gdcurv);
    sj_glob = gd_info_ind_lcext2glphy_j(sj, gdcurv);
    sk_glob = gd_info_ind_lcext2glphy_k(sk, gdcurv);
  }

  // reduce global index and shift values
  int sendbufi = si_glob;
  MPI_Allreduce(&sendbufi, &si_glob, 1, MPI_INT, MPI_MAX, comm);

  sendbufi = sj_glob;
  MPI_Allreduce(&sendbufi, &sj_glob, 1, MPI_INT, MPI_MAX, comm);

  sendbufi = sk_glob;
  MPI_Allreduce(&sendbufi, &sk_glob, 1, MPI_INT, MPI_MAX, comm);

  float sendbuf = sx_inc;
  MPI_Allreduce(&sendbuf, &sx_inc, 1, MPI_FLOAT, MPI_SUM, comm);

  sendbuf = sy_inc;
  MPI_Allreduce(&sendbuf, &sy_inc, 1, MPI_FLOAT, MPI_SUM, comm);

  sendbuf = sz_inc;
  MPI_Allreduce(&sendbuf, &sz_inc, 1, MPI_FLOAT, MPI_SUM, comm);

  *ou_si = si_glob;
  *ou_sj = sj_glob;
  *ou_sk = sk_glob;
  *ou_sx_inc = sx_inc;
  *ou_sy_inc = sy_inc;
  *ou_sz_inc = sz_inc;

  return is_here; 
}

__device__ int
gd_curv_coord_to_glob_indx_gpu(gdcurv_t *gdcurv,
                               float sx,
                               float sy,
                               float sz,
                               MPI_Comm comm,
                               int myid,
                               int   *ou_si, int *ou_sj, int *ou_sk,
                               float *ou_sx_inc, float *ou_sy_inc, float *ou_sz_inc)
{
  int is_here = 0;

  //NOTE si_glob sj_glob sk_glob must less -3. due to ghost points length is 3.
  int si_glob = -1000;
  int sj_glob = -1000;
  int sk_glob = -1000;
  float sx_inc = 0.0;
  float sy_inc = 0.0;
  float sz_inc = 0.0;
  int si = 0;
  int sj = 0;
  int sk = 0;
  // if located in this thread
  is_here = gd_curv_coord_to_local_indx(gdcurv,sx,sy,sz,
                                    &si, &sj, &sk, &sx_inc, &sy_inc, &sz_inc);

  // if in this thread
  if ( is_here == 1)
  {
    // conver to global index
    si_glob = gd_info_ind_lcext2glphy_i(si, gdcurv);
    sj_glob = gd_info_ind_lcext2glphy_j(sj, gdcurv);
    sk_glob = gd_info_ind_lcext2glphy_k(sk, gdcurv);
  }

  *ou_si = si_glob;
  *ou_sj = sj_glob;
  *ou_sk = sk_glob;
  *ou_sx_inc = sx_inc;
  *ou_sy_inc = sy_inc;
  *ou_sz_inc = sz_inc;

  return is_here; 
}

/* 
 * if the nearest point in this thread then search its grid index
 *   return value:
 *      1 - in this thread
 *      0 - not in this thread
 */

__host__ __device__ int
gd_curv_coord_to_local_indx(gdcurv_t *gdcurv,
                            float sx, float sy, float sz,
                            int *si, int *sj, int *sk,
                            float *sx_inc, float *sy_inc, float *sz_inc)
{
  int is_here = 0; // default outside

  // not here if outside coord range
  if ( sx < gdcurv->xmin || sx > gdcurv->xmax ||
       sy < gdcurv->ymin || sy > gdcurv->ymax ||
       sz < gdcurv->zmin || sz > gdcurv->zmax)
  {
    return is_here;
  }

  int nx = gdcurv->nx;
  int ny = gdcurv->ny;
  int nz = gdcurv->nz;
  int ni1 = gdcurv->ni1;
  int ni2 = gdcurv->ni2;
  int nj1 = gdcurv->nj1;
  int nj2 = gdcurv->nj2;
  int nk1 = gdcurv->nk1;
  int nk2 = gdcurv->nk2;
  size_t siz_iy = gdcurv->siz_iy;
  size_t siz_iz = gdcurv->siz_iz;
  
  float *x3d = gdcurv->x3d;
  float *y3d = gdcurv->y3d;
  float *z3d = gdcurv->z3d;
  
  // init closest point
  float min_dist = sqrtf(  (sx - x3d[0]) * (sx - x3d[0])
      + (sy - y3d[0]) * (sy - y3d[0])
      + (sz - z3d[0]) * (sz - z3d[0]) );
  int min_dist_i = 0 ;
  int min_dist_j = 0 ;
  int min_dist_k = 0 ;

  // compute distance to each point
  for (int k=0; k<nz; k++) {
    for (int j=0; j<ny; j++) {
      for (int i=0; i<nx; i++)
      {
        size_t iptr = i + j * siz_iy + k * siz_iz;

        float x = x3d[iptr];
        float y = y3d[iptr];
        float z = z3d[iptr];

        float DistInt = sqrtf(  (sx - x) * (sx - x)
            + (sy - y) * (sy - y)
            + (sz - z) * (sz - z) );

        // replace closest point
        if (min_dist > DistInt)
        {
          min_dist = DistInt;
          min_dist_i = i;
          min_dist_j = j;
          min_dist_k = k;
        }
      }
    }
  }

  // if nearest index is outside phys region, not here
  if ( min_dist_i < ni1 || min_dist_i > ni2 ||
      min_dist_j < nj1 || min_dist_j > nj2 ||
      min_dist_k < nk1 || min_dist_k > nk2 )
  {
    is_here = 0;
    return is_here;
  }

  // in this thread
  is_here = 1;

  float points_x[8];
  float points_y[8];
  float points_z[8];
  float points_i[8];
  float points_j[8];
  float points_k[8];

  for (int kk=0; kk<2; kk++)
  {
    for (int jj=0; jj<2; jj++)
    {
      for (int ii=0; ii<2; ii++)
      {
        int cur_i = min_dist_i-1+ii;
        int cur_j = min_dist_j-1+jj;
        int cur_k = min_dist_k-1+kk;

        for (int n3=0; n3<2; n3++) {
          for (int n2=0; n2<2; n2++) {
            for (int n1=0; n1<2; n1++) {
              int iptr_cube = n1 + n2 * 2 + n3 * 4;
              int iptr = (cur_i+n1) + (cur_j+n2) * siz_iy +
                (cur_k+n3) * siz_iz;
              points_x[iptr_cube] = x3d[iptr];
              points_y[iptr_cube] = y3d[iptr];
              points_z[iptr_cube] = z3d[iptr];
              points_i[iptr_cube] = cur_i+n1;
              points_j[iptr_cube] = cur_j+n2;
              points_k[iptr_cube] = cur_k+n3;
            }
          }
        }
        if (isPointInHexahedron_c(sx,sy,sz,points_x,points_y,points_z) == 1)
        {
          float si_curv, sj_curv, sk_curv;

          gd_curv_coord2index_sample(sx,sy,sz,
              8,
              points_x,points_y,points_z,
              points_i,points_j,points_k,
              100,100,100,
              &si_curv, &sj_curv, &sk_curv);

          // convert to return values
          *si = min_dist_i;
          *sj = min_dist_j;
          *sk = min_dist_k;
          *sx_inc = si_curv - min_dist_i;
          *sy_inc = sj_curv - min_dist_j;
          *sz_inc = sk_curv - min_dist_k;

          return is_here;
        }
      }
    }
  }

  // if not in any cube due to bug, set default value
  //    if everything is right, should be return 10 line before
  *si = min_dist_i;
  *sj = min_dist_j;
  *sk = min_dist_k;
  *sx_inc = 0.0;
  *sy_inc = 0.0;
  *sz_inc = 0.0;

  return is_here;
}


/*
 * convert depth to axis
 */
__host__ __device__
int
gd_curv_depth_to_axis(gdcurv_t *gdcurv,
                      float sx,
                      float sy,
                      float *sz,
                      MPI_Comm comm,
                      int myid)
{
  int ierr = 0;

  // not here if outside coord range
  if ( sx < gdcurv->xmin || sx > gdcurv->xmax ||
       sy < gdcurv->ymin || sy > gdcurv->ymax )
  {
    return ierr;
  }

  float points_x[4];
  float points_y[4];
  float points_z[4];

  size_t iptr_k, iptr_j, iptr;

  // take upper-right cell, thus do not take last index
  int k_tile = GD_TILE_NZ - 1;
  {
    for (int j_tile = 0; j_tile < GD_TILE_NY; j_tile++)
    {
      for (int i_tile = 0; i_tile < GD_TILE_NX; i_tile++)
      {
        int iptr_tile = i_tile + j_tile * GD_TILE_NX + k_tile * GD_TILE_NX * GD_TILE_NY;
        if (  sx < gdcurv->tile_xmin[iptr_tile] ||
              sx > gdcurv->tile_xmax[iptr_tile] ||
              sy < gdcurv->tile_ymin[iptr_tile] ||
              sy > gdcurv->tile_ymax[iptr_tile])
        {
          // loop next tile
          continue;
        }

        // otherwise may in this tile
        int k = gdcurv->tile_kend[k_tile];
        {
          iptr_k = k * gdcurv->siz_iz;
          for (int j = gdcurv->tile_jstart[j_tile]; j <= gdcurv->tile_jend[j_tile]; j++)
          {
            iptr_j = iptr_k + j * gdcurv->siz_iy;
            for (int i = gdcurv->tile_istart[i_tile]; i <= gdcurv->tile_iend[i_tile]; i++)
            {
              iptr = i + iptr_j;

              // use AABB algorith
              if (  sx < gdcurv->cell_xmin[iptr] ||
                    sx > gdcurv->cell_xmax[iptr] ||
                    sy < gdcurv->cell_ymin[iptr] ||
                    sy > gdcurv->cell_ymax[iptr] )
              {
                // loop next cell
                continue;
              }

              // otherwise may in this cell, use inpolygon to check

              // set cell points
              for (int n2=0; n2<2; n2++) {
                for (int n1=0; n1<2; n1++) {
                  int iptr_cube = n1 + n2 * 2;
                  size_t iptr_pt = (i+n1) + (j+n2) * gdcurv->siz_iy + k * gdcurv->siz_iz;
                  points_x[iptr_cube] = gdcurv->x3d[iptr_pt];
                  points_y[iptr_cube] = gdcurv->y3d[iptr_pt];
                  points_z[iptr_cube] = gdcurv->z3d[iptr_pt];
                }
              }

              // interp z if in this cell
              if (fdlib_math_isPoint2InQuad(sx,sy,points_x,points_y) == 1)
              {
                float ztopo = fdlib_math_rdinterp_2d(sx,sy,4,points_x,points_y,points_z);
                 
                *sz = ztopo - (*sz);

                return ierr;
              }
              
            } // i
          } // j
        } // k

      } // i_tile
    } // j_tile
  } // k_tile

  return ierr;
}

/* 
 * find relative coord shift in this cell using sampling
 */

__host__ __device__ int
gd_curv_coord2shift_sample(float sx, float sy, float sz, 
    int num_points,
    float *points_x, // x coord of all points
    float *points_y,
    float *points_z,
    int    nx_sample,
    int    ny_sample,
    int    nz_sample,
    float *si_shift, // interped curv coord
    float *sj_shift,
    float *sk_shift)
{
  float Lx[2], Ly[2], Lz[2];

  // init closest point
  float min_dist = sqrtf(  (sx - points_x[0]) * (sx - points_x[0])
      + (sy - points_y[0]) * (sy - points_y[0])
      + (sz - points_z[0]) * (sz - points_z[0]) );
  int min_dist_i = 0 ;
  int min_dist_j = 0 ;
  int min_dist_k = 0 ;

  // linear interp for all sample
  for (int n3=0; n3<nz_sample+1; n3++)
  {
    Lz[1] = (float)(n3) / (float)(nz_sample);
    Lz[0] = 1.0 - Lz[1];
    for (int n2=0; n2<ny_sample+1; n2++)
    {
      Ly[1] = (float)(n2) / (float)(ny_sample);
      Ly[0] = 1.0 - Ly[1];
      for (int n1=0; n1<nx_sample+1; n1++)
      {
        Lx[1] = (float)(n1) / (float)(nx_sample);
        Lx[0] = 1.0 - Lx[1];

        // interp
        float x_pt=0;
        float y_pt=0;
        float z_pt=0;
        for (int kk=0; kk<2; kk++) {
          for (int jj=0; jj<2; jj++) {
            for (int ii=0; ii<2; ii++)
            {
              int iptr_cube = ii + jj * 2 + kk * 4;
              x_pt += Lx[ii]*Ly[jj]*Lz[kk] * points_x[iptr_cube];
              y_pt += Lx[ii]*Ly[jj]*Lz[kk] * points_y[iptr_cube];
              z_pt += Lx[ii]*Ly[jj]*Lz[kk] * points_z[iptr_cube];
            }
          }
        }

        // find min dist
        float DistInt = sqrtf(  (sx - x_pt) * (sx - x_pt)
            + (sy - y_pt) * (sy - y_pt)
            + (sz - z_pt) * (sz - z_pt) );

        // replace closest point
        if (min_dist > DistInt)
        {
          min_dist = DistInt;
          min_dist_i = n1;
          min_dist_j = n2;
          min_dist_k = n3;
        }
      } // n1
    } // n2
  } // n3

  *si_shift = (float)min_dist_i / (float)nx_sample;
  *sj_shift = (float)min_dist_j / (float)ny_sample;
  *sk_shift = (float)min_dist_k / (float)nz_sample;

  return 0;
}
/* 
 * find curv index using sampling
 */

__host__ __device__ int
gd_curv_coord2index_sample(float sx, float sy, float sz, 
    int num_points,
    float *points_x, // x coord of all points
    float *points_y,
    float *points_z,
    float *points_i, // curv coord of all points
    float *points_j,
    float *points_k,
    int    nx_sample,
    int    ny_sample,
    int    nz_sample,
    float *si_curv, // interped curv coord
    float *sj_curv,
    float *sk_curv)
{
  float Lx[2], Ly[2], Lz[2];
  // init closest point
  float min_dist = sqrtf(  (sx - points_x[0]) * (sx - points_x[0])
      + (sy - points_y[0]) * (sy - points_y[0])
      + (sz - points_z[0]) * (sz - points_z[0]) );
  int min_dist_i = 0 ;
  int min_dist_j = 0 ;
  int min_dist_k = 0 ;

  // linear interp for all sample
  for (int n3=0; n3<nz_sample+1; n3++)
  {
    Lz[1] = (float)(n3) / (float)(nz_sample);
    Lz[0] = 1.0 - Lz[1];
    for (int n2=0; n2<ny_sample+1; n2++)
    {
      Ly[1] = (float)(n2) / (float)(ny_sample);
      Ly[0] = 1.0 - Ly[1];
      for (int n1=0; n1<nx_sample+1; n1++)
      {
        Lx[1] = (float)(n1) / (float)(nx_sample);
        Lx[0] = 1.0 - Lx[1];

        // interp
        float x_pt=0;
        float y_pt=0;
        float z_pt=0;
        for (int kk=0; kk<2; kk++) {
          for (int jj=0; jj<2; jj++) {
            for (int ii=0; ii<2; ii++)
            {
              int iptr_cube = ii + jj * 2 + kk * 4;
              x_pt += Lx[ii]*Ly[jj]*Lz[kk] * points_x[iptr_cube];
              y_pt += Lx[ii]*Ly[jj]*Lz[kk] * points_y[iptr_cube];
              z_pt += Lx[ii]*Ly[jj]*Lz[kk] * points_z[iptr_cube];
            }
          }
        }

        // find min dist
        float DistInt = sqrtf(  (sx - x_pt) * (sx - x_pt)
            + (sy - y_pt) * (sy - y_pt)
            + (sz - z_pt) * (sz - z_pt) );

        // replace closest point
        if (min_dist > DistInt)
        {
          min_dist = DistInt;
          min_dist_i = n1;
          min_dist_j = n2;
          min_dist_k = n3;
        }
      } // n1
    } // n2
  } // n3

  *si_curv = points_i[0] + (float)min_dist_i / (float)nx_sample;
  *sj_curv = points_j[0] + (float)min_dist_j / (float)ny_sample;
  *sk_curv = points_k[0] + (float)min_dist_k / (float)nz_sample;

  return 0;
}

/* 
 * interp curv coord using inverse distance interp
 */

  int
gd_curv_coord2index_rdinterp(float sx, float sy, float sz, 
    int num_points,
    float *points_x, // x coord of all points
    float *points_y,
    float *points_z,
    float *points_i, // curv coord of all points
    float *points_j,
    float *points_k,
    float *si_curv, // interped curv coord
    float *sj_curv,
    float *sk_curv)
{
  float weight[num_points];
  float total_weight = 0.0 ;

  // cal weight
  int at_point_indx = -1;
  for (int i=0; i<num_points; i++)
  {
    float dist = sqrtf ((sx - points_x[i]) * (sx - points_x[i])
        + (sy - points_y[i]) * (sy - points_y[i])
        + (sz - points_z[i]) * (sz - points_z[i])
        );
    if (dist < 1e-9) {
      at_point_indx = i;
    } else {
      weight[i]   = 1.0 / dist;
      total_weight += weight[i];
    }
  }
  // if at a point
  if (at_point_indx > 0) {
    total_weight = 1.0;
    // other weight 0
    for (int i=0; i<num_points; i++) {
      weight[i] = 0.0;
    }
    // point weight 1
    weight[at_point_indx] = 1.0;
  }

  // interp

  *si_curv = 0.0;
  *sj_curv = 0.0;
  *sk_curv = 0.0;

  for (int i=0; i<num_points; i++)
  {
    weight[i] *= 1.0 / total_weight ;

    (*si_curv) += weight[i] * points_i[i];
    (*sj_curv) += weight[i] * points_j[i]; 
    (*sk_curv) += weight[i] * points_k[i];  

    fprintf(stdout,"---- i=%d,weight=%f,points_i=%f,points_j=%f,points_k=%f\n",
        i,weight[i],points_i[i],points_j[i],points_k[i]);
  }

  return 0;
}

float
gd_coord_get_x(gdcurv_t *gdcurv, int i, int j, int k)
{
  float var = 0.0;

  size_t iptr = i + j * gdcurv->siz_iy + k * gdcurv->siz_iz;
  var = gdcurv->x3d[iptr];

  return var;
}

float
gd_coord_get_y(gdcurv_t *gdcurv, int i, int j, int k)
{
  float var = 0.0;

  size_t iptr = i + j * gdcurv->siz_iy + k * gdcurv->siz_iz;
  var = gdcurv->y3d[iptr];

  return var;
}

float
gd_coord_get_z(gdcurv_t *gdcurv, int i, int j, int k)
{
  float var = 0.0;

  size_t iptr = i + j * gdcurv->siz_iy + k * gdcurv->siz_iz;
  var = gdcurv->z3d[iptr];

  return var;
}

/*
 * Input: vx, vy, vz are the EIGHT vertexes of the hexahedron 
 *
 *    ↑ +z       4----6
 *    |         /|   /|
 *             / 0--/-2
 *            5----7 /
 *            |/   |/
 *            1----3
 *
 *
 */
// c++ version is coding by jiangluqian
// c cersion is coding by lihualin
__host__ __device__
int isPointInHexahedron_c(float px,  float py,  float pz,
                          float *vx, float *vy, float *vz)
{
  float point[3] = {px, py, pz};
	/* 
	 * Just for cgfd3D, in which the grid mesh maybe not a hexahedron,
	 */
  // order is back front left right top bottom 
  float hexa[6][3][3] = {
  {{vx[0], vy[0], vz[0]},{vx[4], vy[4], vz[4]},{vx[6], vy[6], vz[6]}},
  {{vx[7], vy[7], vz[7]},{vx[5], vy[5], vz[5]},{vx[1], vy[1], vz[1]}},
  {{vx[5], vy[5], vz[5]},{vx[4], vy[4], vz[4]},{vx[0], vy[0], vz[0]}},
  {{vx[2], vy[2], vz[2]},{vx[6], vy[6], vz[6]},{vx[7], vy[7], vz[7]}},
  {{vx[4], vy[4], vz[4]},{vx[5], vy[5], vz[5]},{vx[7], vy[7], vz[7]}},
  {{vx[3], vy[3], vz[3]},{vx[1], vy[1], vz[1]},{vx[0], vy[0], vz[0]}},
  };

/* 
 * Check whether the point is in the polyhedron.
 * Note: The hexahedron must be convex!
 */
  float sign;
  float len_p2f;
  float p2f[3] = {0};
  float normal_unit[3] = {0};
  for(int i=0; i<6; i++)
  {
    point2face(hexa[i][0],point,p2f); 
    face_normal(hexa[i],normal_unit);
    sign = fdlib_math_dot_product(p2f,normal_unit);
    len_p2f=sqrt(fdlib_math_dot_product(p2f,p2f));
    sign /= len_p2f;
    if(sign < 0.0) return 0;
  }
  return 1;
}

__host__ __device__
int point2face(float *hexa1d,float *point, float *p2f)
{
  for(int i=0; i<3; i++)
  {
    p2f[i] = hexa1d[i] - point[i];
  }
  return 0;
}

__host__ __device__
int face_normal(float (*hexa2d)[3], float *normal_unit)
{
  float A[3];
  float B[3];
  float normal[3]; // normal vector
  float length;
  for(int i=0;i<3;i++)
  {
    A[i] = hexa2d[1][i] - hexa2d[0][i];
    B[i] = hexa2d[2][i] - hexa2d[0][i];
  }
  // calculate normal vector
  fdlib_math_cross_product(A, B, normal);
  // Normalized the normal vector
  length = sqrt(fdlib_math_dot_product(normal, normal));
  for(int i=0; i<3; i++)
  {
    normal_unit[i] = normal[i] / length;
  }

  return 0;
}

//
// set grid size
//

int
gd_info_set(gdcurv_t *const gdcurv,
            const mympi_t *const mympi,
            const int number_of_total_grid_points_x,
            const int number_of_total_grid_points_y,
            const int number_of_total_grid_points_z,
                  int abs_num_of_layers[][2],
            const int fdx_nghosts,
            int const fdy_nghosts,
            const int fdz_nghosts,
            const int verbose)
{
  int ierr = 0;

  // determine ni
  int nx_et = number_of_total_grid_points_x;

  // double cfspml load
  nx_et += abs_num_of_layers[0][0] + abs_num_of_layers[0][1];

  // partition into average plus left at last
  int nx_avg  = nx_et / mympi->nprocx;
  int nx_left = nx_et % mympi->nprocx;

  if (nx_avg < 2 * fdx_nghosts) {
    fprintf(stdout,"should not be less than 2 * fdx_nghosts");
    exit(1);
  }

  if (nx_avg<abs_num_of_layers[0][0] || nx_avg<abs_num_of_layers[0][1]) {
    fprintf(stdout,"should not be less than abs_num_of_layers");
    exit(1);
  }

  // default set to average value
  int ni = nx_avg;
  // subtract nlay for pml node
  if (mympi->neighid[0] == MPI_PROC_NULL) {
    ni -= abs_num_of_layers[0][0];
  }
  if (mympi->neighid[1] == MPI_PROC_NULL) {
    ni -= abs_num_of_layers[0][1];
  }
  // first nx_left node add one more point
  if (mympi->topoid[0] < nx_left) {
    ni++;
  }
  // global index
  if (mympi->topoid[0]==0) {
    gdcurv->gni1 = 0;
  } else {
    gdcurv->gni1 = mympi->topoid[0] * nx_avg - abs_num_of_layers[0][0];
  }
  if (nx_left != 0) {
    gdcurv->gni1 += (mympi->topoid[0] < nx_left) ? mympi->topoid[0] : nx_left;
  }

  // determine nj
  int ny_et = number_of_total_grid_points_y;
  // double cfspml load
  ny_et += abs_num_of_layers[1][0] + abs_num_of_layers[1][1];
  int ny_avg  = ny_et / mympi->nprocy;
  int ny_left = ny_et % mympi->nprocy;

  if (ny_avg < 2 * fdy_nghosts) {
    fprintf(stdout,"should not be less than 2 * fdy_nghosts");
    exit(1);
  }
  if (ny_avg<abs_num_of_layers[1][0] || ny_avg<abs_num_of_layers[1][1]) {
    fprintf(stdout,"should not be less than abs_num_of_layers");
    exit(1);
  }
  int nj = ny_avg;
  if (mympi->neighid[2] == MPI_PROC_NULL) {
    nj -= abs_num_of_layers[1][0];
  }
  if (mympi->neighid[3] == MPI_PROC_NULL) {
    nj -= abs_num_of_layers[1][1];
  }
  // not equal divided points given to first ny_left procs
  if (mympi->topoid[1] < ny_left) {
    nj++;
  }
  // global index
  if (mympi->topoid[1]==0) {
    gdcurv->gnj1 = 0;
  } else {
    gdcurv->gnj1 = mympi->topoid[1] * ny_avg - abs_num_of_layers[1][0];
  }
  if (ny_left != 0) {
    gdcurv->gnj1 += (mympi->topoid[1] < ny_left) ? mympi->topoid[1] : ny_left;
  }

  // determine nk
  int nz_et = number_of_total_grid_points_z;
  // double cfspml load
  // if has free_surface, abs_num_of_layers[2][1] = 0
  nz_et += abs_num_of_layers[2][0] + abs_num_of_layers[2][1];
  int nz_avg  = nz_et / mympi->nprocz;
  int nz_left = nz_et % mympi->nprocz;
  if (nz_avg < 2 * fdz_nghosts) {
    fprintf(stdout,"should not be less than 2 * fdz_nghosts");
    exit(1);
  }
  if (nz_avg<abs_num_of_layers[2][0] || nz_avg<abs_num_of_layers[2][1]) {
    fprintf(stdout,"should not be less than abs_num_of_layers");
    exit(1);
  }
  int nk = nz_avg;
  if (mympi->neighid[4] == MPI_PROC_NULL) {
    nk -= abs_num_of_layers[2][0];
  }
  if (mympi->neighid[5] == MPI_PROC_NULL) {
    nk -= abs_num_of_layers[2][1];
  }
  // not equal divided points given to first nz_left procs
  if (mympi->topoid[2] < nz_left) {
    nk++;
  }
  // global index
  if (mympi->topoid[2]==0) {
    gdcurv->gnk1 = 0;
  } else {
    gdcurv->gnk1 = mympi->topoid[2] * nz_avg - abs_num_of_layers[2][0];
  }
  if (nz_left != 0) {
    gdcurv->gnk1 += (mympi->topoid[2] < nz_left) ? mympi->topoid[2] : nz_left;
  }
  
  // add ghost points
  int nx = ni + 2 * fdx_nghosts;
  int ny = nj + 2 * fdy_nghosts;
  int nz = nk + 2 * fdz_nghosts;

  gdcurv->ni = ni;
  gdcurv->nj = nj;
  gdcurv->nk = nk;

  gdcurv->nx = nx;
  gdcurv->ny = ny;
  gdcurv->nz = nz;

  gdcurv->ni1 = fdx_nghosts;
  gdcurv->ni2 = gdcurv->ni1 + ni - 1;

  gdcurv->nj1 = fdy_nghosts;
  gdcurv->nj2 = gdcurv->nj1 + nj - 1;

  gdcurv->nk1 = fdz_nghosts;
  gdcurv->nk2 = gdcurv->nk1 + nk - 1;

  // global index end
  gdcurv->gni2 = gdcurv->gni1 + gdcurv->ni - 1;
  gdcurv->gnj2 = gdcurv->gnj1 + gdcurv->nj - 1;
  gdcurv->gnk2 = gdcurv->gnk1 + gdcurv->nk - 1;

  gdcurv->ni1_to_glob_phys0 = gdcurv->gni1;
  gdcurv->ni2_to_glob_phys0 = gdcurv->gni2;
  gdcurv->nj1_to_glob_phys0 = gdcurv->gnj1;
  gdcurv->nj2_to_glob_phys0 = gdcurv->gnj2;
  gdcurv->nk1_to_glob_phys0 = gdcurv->gnk1;
  gdcurv->nk2_to_glob_phys0 = gdcurv->gnk2;
  
  gdcurv->npoint_x = number_of_total_grid_points_x; 
  gdcurv->npoint_y = number_of_total_grid_points_y; 
  gdcurv->npoint_z = number_of_total_grid_points_z;

  // x dimention varies first
  gdcurv->siz_iy   = nx; 
  gdcurv->siz_iz   = nx * ny; 
  gdcurv->siz_icmp = gdcurv->siz_iz * nz;

  gdcurv->siz_slice_yz = ny * nz;
  gdcurv->siz_slice_yz2 = 2 * ny * nz;

  // set npoint_ghosts according to fdz_nghosts
  gdcurv->npoint_ghosts = fdz_nghosts;

  gdcurv->fdx_nghosts = fdx_nghosts;
  gdcurv->fdy_nghosts = fdy_nghosts;
  gdcurv->fdz_nghosts = fdz_nghosts;

  gdcurv->index_name = fdlib_mem_malloc_2l_char(
                        CONST_NDIM, CONST_MAX_STRLEN, "gdcurv name");

  // grid coord name
  sprintf(gdcurv->index_name[0],"%s","i");
  sprintf(gdcurv->index_name[1],"%s","j");
  sprintf(gdcurv->index_name[2],"%s","k");

  return ierr;
}

/*
 * give a local index ref, check if in this thread
 */

int
gd_info_lindx_is_inner(int i, int j, int k, gdcurv_t *gdcurv)
{
  int is_in = 0;

  if (   i >= gdcurv->ni1 && i <= gdcurv->ni2
      && j >= gdcurv->nj1 && j <= gdcurv->nj2
      && k >= gdcurv->nk1 && k <= gdcurv->nk2)
  {
    is_in = 1;
  }

  return is_in;
}  

/*
 * give a global index ref to phys0, check if in this thread
 */

int
gd_info_gindx_is_inner(int gi, int gj, int gk, gdcurv_t *gdcurv)
{
  int ishere = 0;

  if ( gi >= gdcurv->ni1_to_glob_phys0 && gi <= gdcurv->ni2_to_glob_phys0 &&
       gj >= gdcurv->nj1_to_glob_phys0 && gj <= gdcurv->nj2_to_glob_phys0 &&
       gk >= gdcurv->nk1_to_glob_phys0 && gk <= gdcurv->nk2_to_glob_phys0 )
  {
    ishere = 1;
  }

  return ishere;
}

/*
 * glphyinx, glextind, gp,ge
 * lcphyind, lcextind
 * gl: global
 * lc: local
 * inx: index
 * phy: physical points only, do not count ghost
 * ext: include extended points, with ghots points
 */

int
gd_info_gindx_is_inner_i(int gi, gdcurv_t *gdcurv)
{
  int ishere = 0;

  if ( gi >= gdcurv->ni1_to_glob_phys0 && gi <= gdcurv->ni2_to_glob_phys0)
  {
    ishere = 1;
  }

  return ishere;
}

int
gd_info_gindx_is_inner_j(int gj, gdcurv_t *gdcurv)
{
  int ishere = 0;

  if ( gj >= gdcurv->nj1_to_glob_phys0 && gj <= gdcurv->nj2_to_glob_phys0)
  {
    ishere = 1;
  }

  return ishere;
}

int
gd_info_gindx_is_inner_k(int gk, gdcurv_t *gdcurv)
{
  int ishere = 0;

  if ( gk >= gdcurv->nk1_to_glob_phys0 && gk <= gdcurv->nk2_to_glob_phys0)
  {
    ishere = 1;
  }

  return ishere;
}

/*
 * convert global index to local
 */

int
gd_info_ind_glphy2lcext_i(int gi, gdcurv_t *gdcurv)
{
  return gi - gdcurv->ni1_to_glob_phys0 + gdcurv->npoint_ghosts;
}

int
gd_info_ind_glphy2lcext_j(int gj, gdcurv_t *gdcurv)
{
  return gj - gdcurv->nj1_to_glob_phys0 + gdcurv->npoint_ghosts;
}

int
gd_info_ind_glphy2lcext_k(int gk, gdcurv_t *gdcurv)
{
  return gk - gdcurv->nk1_to_glob_phys0 + gdcurv->npoint_ghosts;
}

/*
 * convert local index to global
 */

__host__ __device__ int
gd_info_ind_lcext2glphy_i(int i, gdcurv_t *gdcurv)
{
  return i - gdcurv->npoint_ghosts + gdcurv->ni1_to_glob_phys0;
}

__host__ __device__ int
gd_info_ind_lcext2glphy_j(int j, gdcurv_t *gdcurv)
{
  return j - gdcurv->npoint_ghosts + gdcurv->nj1_to_glob_phys0;
}

__host__ __device__ int
gd_info_ind_lcext2glphy_k(int k, gdcurv_t *gdcurv)
{
  return k - gdcurv->npoint_ghosts + gdcurv->nk1_to_glob_phys0;
}

/*
 * print for QC
 */

int
gd_info_print(gdcurv_t *gdcurv)
{    
  fprintf(stdout, "-------------------------------------------------------\n");
  fprintf(stdout, "--> grid info:\n");
  fprintf(stdout, "-------------------------------------------------------\n");
  fprintf(stdout, " nx    = %-10d\n", gdcurv->nx);
  fprintf(stdout, " ny    = %-10d\n", gdcurv->ny);
  fprintf(stdout, " nz    = %-10d\n", gdcurv->nz);
  fprintf(stdout, " ni    = %-10d\n", gdcurv->ni);
  fprintf(stdout, " nj    = %-10d\n", gdcurv->nj);
  fprintf(stdout, " nk    = %-10d\n", gdcurv->nk);

  fprintf(stdout, " ni1   = %-10d\n", gdcurv->ni1);
  fprintf(stdout, " ni2   = %-10d\n", gdcurv->ni2);
  fprintf(stdout, " nj1   = %-10d\n", gdcurv->nj1);
  fprintf(stdout, " nj2   = %-10d\n", gdcurv->nj2);
  fprintf(stdout, " nk1   = %-10d\n", gdcurv->nk1);
  fprintf(stdout, " nk2   = %-10d\n", gdcurv->nk2);

  fprintf(stdout, " ni1_to_glob_phys0   = %-10d\n", gdcurv->gni1);
  fprintf(stdout, " ni2_to_glob_phys0   = %-10d\n", gdcurv->gni2);
  fprintf(stdout, " nj1_to_glob_phys0   = %-10d\n", gdcurv->gnj1);
  fprintf(stdout, " nj2_to_glob_phys0   = %-10d\n", gdcurv->gnj2);
  fprintf(stdout, " nk1_to_glob_phys0   = %-10d\n", gdcurv->gnk1);
  fprintf(stdout, " nk2_to_glob_phys0   = %-10d\n", gdcurv->gnk2);

  fprintf(stdout, "\n-------------------------------------------------------\n");
  fprintf(stdout, "print grid structure info:\n");
  fprintf(stdout, "-------------------------------------------------------\n\n");

  fprintf(stdout," xmin=%g, xmax=%g\n", gdcurv->xmin,gdcurv->xmax);
  fprintf(stdout," ymin=%g, ymax=%g\n", gdcurv->ymin,gdcurv->ymax);
  fprintf(stdout," zmin=%g, zmax=%g\n", gdcurv->zmin,gdcurv->zmax);
  /*
  for (int k_tile = 0; k_tile < GD_TILE_NZ; k_tile++)
  {
    fprintf(stdout," tile k=%d, pt k in (%d,%d)\n",
                k_tile, gdcurv->tile_kstart[k_tile],gdcurv->tile_kend[k_tile]);
  }
  for (int j_tile = 0; j_tile < GD_TILE_NY; j_tile++)
  {
    fprintf(stdout," tile j=%d, pt j in (%d,%d)\n",
                  j_tile, gdcurv->tile_jstart[j_tile],gdcurv->tile_jend[j_tile]);
  }
  for (int i_tile = 0; i_tile < GD_TILE_NX; i_tile++)
  {
    fprintf(stdout," tile i=%d, pt i in (%d,%d)\n",
                  i_tile, gdcurv->tile_istart[i_tile],gdcurv->tile_iend[i_tile]);
  }
  for (int k_tile = 0; k_tile < GD_TILE_NZ; k_tile++)
  {
    for (int j_tile = 0; j_tile < GD_TILE_NY; j_tile++)
    {
      for (int i_tile = 0; i_tile < GD_TILE_NX; i_tile++)
      {
        int iptr_tile = i_tile + j_tile * GD_TILE_NX + k_tile * GD_TILE_NX * GD_TILE_NY;
        fprintf(stdout," tile %d,%d,%d, range (%g,%g,%g,%g,%g,%g)\n",
                i_tile,j_tile,k_tile,
                gdcurv->tile_xmin[iptr_tile],
                gdcurv->tile_xmax[iptr_tile],
                gdcurv->tile_ymin[iptr_tile],
                gdcurv->tile_ymax[iptr_tile],
                gdcurv->tile_zmin[iptr_tile],
                gdcurv->tile_zmax[iptr_tile]);
      }
    }
  }
  */

  fflush(stdout);

  return(0);
}
