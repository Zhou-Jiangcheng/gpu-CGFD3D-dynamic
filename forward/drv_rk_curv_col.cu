#include "hip/hip_runtime.h"
/*******************************************************************************
 * solver of isotropic elastic 1st-order eqn using curv grid and macdrp schem
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <mpi.h>

#include "fdlib_mem.h"
#include "fdlib_math.h"
#include "blk_t.h"
#include "drv_rk_curv_col.h"
#include "sv_curv_col_el_iso_gpu.h"
#include "sv_curv_col_el_iso_fault_gpu.h"
#include "trial_slipweakening.h"
#include "transform.h"
#include "fault_wav_t.h"
#include "alloc.h"
#include "cuda_common.h"

/*******************************************************************************
 * one simulation over all time steps, could be used in imaging or inversion
 *  simple MPI exchange without computing-communication overlapping
 ******************************************************************************/

int
drv_rk_curv_col_allstep(
  fd_t         *fd,
  gdcurv_t     *gdcurv,
  gd_metric_t  *metric,
  md_t         *md,
  par_t        *par,
  bdryfree_t   *bdryfree,
  bdrypml_t    *bdrypml,
  bdryexp_t    *bdryexp,
  wav_t        *wav,
  mympi_t      *mympi,
  fault_coef_t *fault_coef,
  fault_t      *fault,
  fault_wav_t  *fault_wav,
  iorecv_t     *iorecv,
  ioline_t     *ioline,
  iofault_t    *iofault,
  ioslice_t    *ioslice,
  iosnap_t     *iosnap,
  // time
  float dt, int nt_total, float t0,
  char *output_fname_part,
  char *output_dir,
  const int verbose)
{
  int imethod = par->imethod;
  int io_time_skip = par->io_time_skip;
  int qc_check_nan_number_of_step = par->qc_check_nan_number_of_step;

  int num_rk_stages = fd->num_rk_stages;
  int num_of_pairs =  fd->num_of_pairs;
  float *rk_a = fd->rk_a;
  float *rk_b = fd->rk_b;
  
  //gdcurv
  int ni = gdcurv->ni;
  int nj = gdcurv->nj;
  int nk = gdcurv->nk;
  // fault x index with ghost
  int i0 = par->fault_x_index[0] + gdcurv->fdx_nghosts;
  // mpi
  int myid = mympi->myid;
  int *topoid = mympi->topoid;
  MPI_Comm comm = mympi->comm;
  int *neighid_d = init_neighid_device(mympi->neighid);

  gdcurv_t     gdcurv_d;
  md_t         md_d;
  wav_t        wav_d;
  fd_device_t  fd_device_d;
  gd_metric_t  metric_d;
  fault_coef_t fault_coef_d;
  fault_t      fault_d;
  fault_wav_t  fault_wav_d;
  bdryfree_t   bdryfree_d;
  bdrypml_t    bdrypml_d;
  bdryexp_t    bdryexp_d;

  // init device struct, and copy data from host to device
  init_gdcurv_device(gdcurv, &gdcurv_d);
  init_md_device(md, &md_d);
  init_fd_device(fd, &fd_device_d);
  init_metric_device(metric, &metric_d);
  init_wave_device(wav, &wav_d);
  init_fault_coef_device(gdcurv, fault_coef, &fault_coef_d);
  init_fault_device(gdcurv, fault, &fault_d);
  init_fault_wav_device(fault_wav, &fault_wav_d);
  init_bdryfree_device(gdcurv, bdryfree, &bdryfree_d);
  init_bdrypml_device(gdcurv, bdrypml, &bdrypml_d);
  init_bdryexp_device(gdcurv, bdryexp, &bdryexp_d);

  // get device wavefield 
  float *w_buff = wav->v5d; // size number is V->siz_icmp * (V->ncmp+6)
  // GPU local pointer
  float *w_cur_d;
  float *w_pre_d;
  float *w_rhs_d;
  float *w_end_d;
  float *w_tmp_d;

  float *f_cur_d;
  float *f_pre_d;
  float *f_rhs_d;
  float *f_end_d;
  float *f_tmp_d;

  // get wavefield
  w_pre_d = wav_d.v5d + wav_d.siz_ilevel * 0; // previous level at n
  w_tmp_d = wav_d.v5d + wav_d.siz_ilevel * 1; // intermidate value
  w_rhs_d = wav_d.v5d + wav_d.siz_ilevel * 2; // for rhs
  w_end_d = wav_d.v5d + wav_d.siz_ilevel * 3; // end level at n+1

  // get fault wavefield
  f_pre_d = fault_wav_d.v5d + fault_wav_d.siz_ilevel * 0; // previous level at n
  f_tmp_d = fault_wav_d.v5d + fault_wav_d.siz_ilevel * 1; // intermidate value
  f_rhs_d = fault_wav_d.v5d + fault_wav_d.siz_ilevel * 2; // for rhs
  f_end_d = fault_wav_d.v5d + fault_wav_d.siz_ilevel * 3; // end level at n+1

  int   ipair, istage;
  float t_cur;
  float t_end; // time after this loop for nc output
  // for mpi message
  int   ipair_mpi, istage_mpi;
  // create fault slice nc output files
  if (myid==0 && verbose>0) fprintf(stdout,"prepare fault slice nc output ...\n"); 
  iofault_nc_t iofault_nc;
  io_fault_nc_create(iofault,
                     gdcurv->ni, gdcurv->nj, gdcurv->nk, topoid,
                     &iofault_nc);
  // create slice nc output files
  if (myid==0 && verbose>0) fprintf(stdout,"prepare slice nc output ...\n"); 
  ioslice_nc_t ioslice_nc;
  io_slice_nc_create(ioslice, wav->ncmp, wav->cmp_name,
                     gdcurv->ni, gdcurv->nj, gdcurv->nk, topoid,
                     &ioslice_nc);
  // create snapshot nc output files
  if (myid==0 && verbose>0) fprintf(stdout,"prepare snap nc output ...\n"); 
  iosnap_nc_t  iosnap_nc;
  io_snap_nc_create(iosnap, &iosnap_nc, topoid);

  // only y/z mpi
  int num_of_r_reqs = 8;
  int num_of_s_reqs = 8;

  // set pml for rk
  if(bdrypml_d.is_enable_pml == 1)
  {
    for (int idim=0; idim<CONST_NDIM; idim++) {
      for (int iside=0; iside<2; iside++) {
        if (bdrypml_d.is_sides_pml[idim][iside]==1) {
          bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
          auxvar_d->pre = auxvar_d->var + auxvar_d->siz_ilevel * 0;
          auxvar_d->tmp = auxvar_d->var + auxvar_d->siz_ilevel * 1;
          auxvar_d->rhs = auxvar_d->var + auxvar_d->siz_ilevel * 2;
          auxvar_d->end = auxvar_d->var + auxvar_d->siz_ilevel * 3;
        }
      }
    }
  }

  int isfree = bdryfree_d.is_sides_free[CONST_NDIM-1][1];
  // alloc free surface PGV, PGA and PGD
  float *PG_d = NULL;
  float *PG   = NULL;
  // Dis_accu is Displacemen accumulation, be uesd for PGD calculaton.
  float *Dis_accu_d   = NULL;
  if (isfree == 1)
  {
    PG_d = init_PGVAD_device(gdcurv);
    Dis_accu_d = init_Dis_accu_device(gdcurv);
    PG = (float *) fdlib_mem_calloc_1d_float(CONST_NDIM_5*gdcurv->ny*gdcurv->nx,0.0,"PGV,A,D malloc");
  }
  // calculate conversion matrix for free surface
  if (isfree == 1)
  {
    if (md_d.medium_type == CONST_MEDIUM_ELASTIC_ISO)
    {
      dim3 block(16,16);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_curv_col_el_iso_dvh2dvz_gpu <<<grid, block>>> (gdcurv_d,metric_d,md_d,bdryfree_d,verbose);
      CUDACHECK(hipDeviceSynchronize());
    }
    else
    {
      fprintf(stderr,"ERROR: conversion matrix for medium_type=%d is not implemented\n",
                    md->medium_type);
      MPI_Abort(MPI_COMM_WORLD,1);
    }
  }

  //--------------------------------------------------------
  // time loop
  //--------------------------------------------------------

  if (myid==0 && verbose>0) fprintf(stdout,"start time loop ...\n"); 

  for (int it=0; it<nt_total; it++)
  {
    t_cur = it * dt + t0;
    t_end = t_cur +dt;

    if (myid==0 && verbose>10) fprintf(stdout,"-> it=%d, t=%f\n", it, t_cur);

    // mod to get ipair
    ipair = it % num_of_pairs;
    if (myid==0 && verbose>10) fprintf(stdout, " --> ipair=%d\n",ipair);

    // loop RK stages for one step
    for (istage=0; istage<num_rk_stages; istage++)
    {
      if (myid==0 && verbose>10) fprintf(stdout, " --> istage=%d\n",istage);

      // for mesg
      if (istage != num_rk_stages-1) {
        ipair_mpi = ipair;
        istage_mpi = istage + 1;
      } else {
        ipair_mpi = (it + 1) % num_of_pairs;
        istage_mpi = 0; 
      }

      // use pointer to avoid 1 copy for previous level value
      if (istage==0) {
        w_cur_d = w_pre_d;
        f_cur_d = f_pre_d;
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              bdrypml_d.auxvar[idim][iside].cur = bdrypml_d.auxvar[idim][iside].pre;
            }
          }
        }
      }
      else
      {
        w_cur_d = w_tmp_d;
        f_cur_d = f_tmp_d;
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              bdrypml_d.auxvar[idim][iside].cur = bdrypml_d.auxvar[idim][iside].tmp;
            }
          }
        }
      }

      // compute rhs
      switch (md_d.medium_type)
      {
        case CONST_MEDIUM_ELASTIC_ISO : {

          wave2fault_onestage(
                        w_cur_d, w_rhs_d, wav_d, 
                        f_cur_d, f_rhs_d, fault_wav_d,
                        i0, fault_d, metric_d, gdcurv_d);

          trial_slipweakening_onestage(
                        w_cur_d, f_cur_d, f_pre_d, 
                        i0, isfree, dt,
                        gdcurv_d, metric_d, wav_d, 
                        fault_wav_d, fault_d, fault_coef_d,
                        fd->pair_fdy_op[ipair][istage],
                        fd->pair_fdz_op[ipair][istage],
                        myid, verbose);

          fault2wave_onestage(
                        w_cur_d, wav_d, 
                        f_cur_d, fault_wav_d,
                        i0, fault_d, metric_d, gdcurv_d);

          sv_curv_col_el_iso_onestage(
                        w_cur_d, w_rhs_d, wav_d, gdcurv_d, fd_device_d, 
                        metric_d, md_d, bdryfree_d, bdrypml_d, 
                        fd->pair_fdx_op[ipair][istage],
                        fd->pair_fdy_op[ipair][istage],
                        fd->pair_fdz_op[ipair][istage],
                        myid, verbose);

          sv_curv_col_el_iso_fault_onestage(
                        w_cur_d, w_rhs_d, f_cur_d, f_rhs_d,
                        i0, isfree, imethod, wav_d, 
                        fault_wav_d, fault_d, fault_coef_d,
                        gdcurv_d, metric_d, md_d, bdryfree_d,  
                        fd->pair_fdx_op[ipair][istage],
                        fd->pair_fdy_op[ipair][istage],
                        fd->pair_fdz_op[ipair][istage],
                        myid, verbose);

          break;
        }
      //  synchronize onestage device func.
      CUDACHECK(hipDeviceSynchronize());
      }

      // recv mesg
      MPI_Startall(num_of_r_reqs, mympi->pair_r_reqs[ipair_mpi][istage_mpi]);

      // rk start
      if (istage==0)
      {
        float coef_a = rk_a[istage] * dt;
        float coef_b = rk_b[istage] * dt;

        // temp wavefield
        {
          dim3 block(256);
          dim3 grid;
          grid.x = (wav_d.siz_ilevel + block.x - 1) / block.x;
          wav_update <<<grid, block>>> (wav_d.siz_ilevel, coef_a, w_tmp_d, w_pre_d, w_rhs_d);
        }
        {
          dim3 block(4,8,8);
          dim3 grid;
          grid.x = (2*fault_wav->ncmp + block.x - 1) / block.x;
          grid.y = (nj + block.y - 1) / block.y;
          grid.z = (nk + block.z - 1) / block.z;
          fault_wav_update <<<grid, block>>> (gdcurv_d, fault_wav->ncmp, coef_a, 
                                              fault_d, f_tmp_d, f_pre_d, f_rhs_d);

          fault2wave_onestage(
                        w_tmp_d, wav_d, 
                        f_tmp_d, fault_wav_d,
                        i0, fault_d, metric_d, gdcurv_d);
        }

        // pack and isend
        blk_macdrp_pack_mesg_gpu(w_tmp_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, wav->ncmp, myid);
        blk_macdrp_pack_fault_mesg_gpu(f_tmp_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, fault_wav->ncmp, myid);

        MPI_Startall(num_of_s_reqs, mympi->pair_s_reqs[ipair_mpi][istage_mpi]);
        
        // pml_tmp
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              if (bdrypml_d.is_sides_pml[idim][iside]==1) {
                bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
                dim3 block(256);
                dim3 grid;
                grid.x = (auxvar_d->siz_ilevel + block.x - 1) / block.x;
                wav_update <<<grid, block>>> (
                           auxvar_d->siz_ilevel, coef_a, auxvar_d->tmp, auxvar_d->pre, auxvar_d->rhs);
              }
            }
          }
        }
        // w_end
        {
          dim3 block(256);
          dim3 grid;
          grid.x = (wav_d.siz_ilevel + block.x - 1) / block.x;
          wav_update <<<grid, block>>> (wav_d.siz_ilevel, coef_b, w_end_d, w_pre_d, w_rhs_d);
        }
        {
          dim3 block(4,8,8);
          dim3 grid;
          grid.x = (2*fault_wav->ncmp + block.x - 1) / block.x;
          grid.y = (nj + block.y - 1) / block.y;
          grid.z = (nk + block.z - 1) / block.z;
          fault_wav_update <<<grid, block>>> (gdcurv_d, fault_wav->ncmp, coef_b, 
                                              fault_d, f_end_d, f_pre_d, f_rhs_d);
        }
        {
          dim3 block(8,8);
          dim3 grid;
          grid.x = (nj + block.x - 1) / block.x;
          grid.y = (nk + block.y - 1) / block.y;

          float coef = coef_b / dt;
          fault_stress_update_first <<<grid, block>>> (nj, nk, coef, fault_d);
        }
        // pml_end
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              if (bdrypml_d.is_sides_pml[idim][iside]==1) {
                bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
                dim3 block(256);
                dim3 grid;
                grid.x = (auxvar_d->siz_ilevel + block.x - 1) / block.x;
                wav_update <<<grid, block>>> (
                            auxvar_d->siz_ilevel, coef_b, auxvar_d->end, auxvar_d->pre, auxvar_d->rhs);
              }
            }
          }
        }
      }
      else if (istage<num_rk_stages-1)
      {
        float coef_a = rk_a[istage] * dt;
        float coef_b = rk_b[istage] * dt;
        //temp wavefield
        {
          dim3 block(256);
          dim3 grid;
          grid.x = (wav_d.siz_ilevel + block.x - 1) / block.x;
          wav_update <<<grid, block>>> (wav_d.siz_ilevel, coef_a, w_tmp_d, w_pre_d, w_rhs_d);
          //CUDACHECK(hipDeviceSynchronize());
        }
        {
          dim3 block(4,8,8);
          dim3 grid;
          grid.x = (2*fault_wav->ncmp + block.x - 1) / block.x;
          grid.y = (nj + block.y - 1) / block.y;
          grid.z = (nk + block.z - 1) / block.z;
          fault_wav_update <<<grid, block>>> (gdcurv_d, fault_wav->ncmp, coef_a, 
                                              fault_d, f_tmp_d, f_pre_d, f_rhs_d);
          fault2wave_onestage(
                        w_tmp_d, wav_d, 
                        f_tmp_d, fault_wav_d,
                        i0, fault_d, metric_d, gdcurv_d);
        }

        // pack and isend
        blk_macdrp_pack_mesg_gpu(w_tmp_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, wav->ncmp, myid);
        blk_macdrp_pack_fault_mesg_gpu(f_tmp_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, fault_wav->ncmp, myid);
        MPI_Startall(num_of_s_reqs, mympi->pair_s_reqs[ipair_mpi][istage_mpi]);
        // pml_tmp
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              if (bdrypml_d.is_sides_pml[idim][iside]==1) {
                bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
                dim3 block(256);
                dim3 grid;
                grid.x = (auxvar_d->siz_ilevel + block.x - 1) / block.x;
                wav_update <<<grid, block>>> (
                           auxvar_d->siz_ilevel, coef_a, auxvar_d->tmp, auxvar_d->pre, auxvar_d->rhs);
              }
            }
          }
        }
        // w_end
        {
          dim3 block(256);
          dim3 grid;
          grid.x = (wav_d.siz_ilevel + block.x - 1) / block.x;
          wav_update_end <<<grid, block>>> (wav_d.siz_ilevel, coef_b, w_end_d, w_rhs_d);
        }
        {
          dim3 block(4,8,8);
          dim3 grid;
          grid.x = (2*fault_wav->ncmp + block.x - 1) / block.x;
          grid.y = (nj + block.y - 1) / block.y;
          grid.z = (nk + block.z - 1) / block.z;
          fault_wav_update_end <<<grid, block>>> (gdcurv_d, fault_wav->ncmp, coef_b, 
                                                  fault_d, f_end_d, f_rhs_d);
        }
        {
          dim3 block(8,8);
          dim3 grid;
          grid.x = (nj + block.x - 1) / block.x;
          grid.y = (nk + block.y - 1) / block.y;

          float coef = coef_b / dt;
          fault_stress_update <<<grid, block>>> (nj, nk, coef, fault_d);
        }
        // pml_end
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              if (bdrypml_d.is_sides_pml[idim][iside]==1) {
                bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
                dim3 block(256);
                dim3 grid;
                grid.x = (auxvar_d->siz_ilevel + block.x - 1) / block.x;
                wav_update_end <<<grid, block>>> (
                           auxvar_d->siz_ilevel, coef_b, auxvar_d->end, auxvar_d->rhs);
              }
            }
          }
        }
      }
      else // last stage
      {
        float coef_b = rk_b[istage] * dt;

        // w_end
        {
          dim3 block(256);
          dim3 grid;
          grid.x = (wav_d.siz_ilevel + block.x - 1) / block.x;
          wav_update_end <<<grid, block>>>(wav_d.siz_ilevel, coef_b, w_end_d, w_rhs_d);
        }
        {
          dim3 block(4,8,8);
          dim3 grid;
          grid.x = (2*fault_wav->ncmp + block.x - 1) / block.x;
          grid.y = (nj + block.y - 1) / block.y;
          grid.z = (nk + block.z - 1) / block.z;
          fault_wav_update_end <<<grid, block>>> (gdcurv_d, fault_wav->ncmp, coef_b, 
                                                  fault_d, f_end_d, f_rhs_d);
          fault2wave_onestage(
                        w_end_d, wav_d, 
                        f_end_d, fault_wav_d,
                        i0, fault_d, metric_d, gdcurv_d);
        }
        {
          dim3 block(8,8);
          dim3 grid;
          grid.x = (nj + block.x - 1) / block.x;
          grid.y = (nk + block.y - 1) / block.y;

          float coef = coef_b / dt;
          fault_stress_update <<<grid, block>>> (nj, nk, coef, fault_d);
        }
        
        // pack and isend
        blk_macdrp_pack_mesg_gpu(w_end_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, wav->ncmp, myid);
        blk_macdrp_pack_fault_mesg_gpu(f_end_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, fault_wav->ncmp, myid);
        MPI_Startall(num_of_s_reqs, mympi->pair_s_reqs[ipair_mpi][istage_mpi]);
        // pml_end
        if(bdrypml_d.is_enable_pml == 1)
        {
          for (int idim=0; idim<CONST_NDIM; idim++) {
            for (int iside=0; iside<2; iside++) {
              if (bdrypml->is_sides_pml[idim][iside]==1) {
                bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
                dim3 block(256);
                dim3 grid;
                grid.x = (auxvar_d->siz_ilevel + block.x - 1) / block.x;
                wav_update_end <<<grid, block>>> (
                           auxvar_d->siz_ilevel, coef_b, auxvar_d->end, auxvar_d->rhs);
              }
            }
          }
        }
      }

      MPI_Waitall(num_of_s_reqs, mympi->pair_s_reqs[ipair_mpi][istage_mpi], MPI_STATUS_IGNORE);
      MPI_Waitall(num_of_r_reqs, mympi->pair_r_reqs[ipair_mpi][istage_mpi], MPI_STATUS_IGNORE);
 
      if (istage != num_rk_stages-1) 
      {
        blk_macdrp_unpack_mesg_gpu(w_tmp_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, wav->ncmp, neighid_d);
        blk_macdrp_unpack_fault_mesg_gpu(f_tmp_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, fault_wav->ncmp, neighid_d);
      } else 
      {
        blk_macdrp_unpack_mesg_gpu(w_end_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, wav->ncmp,neighid_d);
        blk_macdrp_unpack_fault_mesg_gpu(f_end_d, fd, gdcurv, mympi, ipair_mpi, istage_mpi, fault_wav->ncmp, neighid_d);
      }
    } // RK stages

    //--------------------------------------------
    // QC
    //--------------------------------------------
    if (qc_check_nan_number_of_step >0  && (it % qc_check_nan_number_of_step) == 0) {
      if (myid==0 && verbose>10) fprintf(stdout,"-> check value nan\n");
        //wav_check_value(w_end);
    }
    //--------------------------------------------
    if (bdryexp_d.is_enable_ablexp == 1) {
      bdry_ablexp_apply(bdryexp_d, gdcurv, w_end_d, wav->ncmp);
    }

    //--------------------------------------------
    // save results
    //--------------------------------------------
    // calculate PGV, PGA and PGD for surface 
    if (isfree == 1)
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni + block.x - 1) / block.x;
      grid.y = (nj + block.y - 1) / block.y;
      PG_calcu_gpu<<<grid, block>>> (w_end_d, w_pre_d, gdcurv_d, PG_d, Dis_accu_d, dt);
    }

    // calculate fault slip, Vs, ... at each dt  
    fault_var_update(f_end_d, it, dt, gdcurv_d, fault_d, fault_coef_d, fault_wav_d);

    //-- recv by interp
    io_recv_keep(iorecv, w_end_d, w_buff, it, wav->ncmp, wav->siz_icmp);

    //-- line values
    io_line_keep(ioline, w_end_d, w_buff, it, wav->ncmp, wav->siz_icmp);
    if((it+1)%io_time_skip == 0)
    {
      int it_skip = (int)(it/io_time_skip);
      // io fault var each dt, use w_buff as buff
      io_fault_nc_put(&iofault_nc, gdcurv, fault_d, w_buff, it_skip, t_end);
      // write slice, use w_buff as buff
      io_slice_nc_put(ioslice,&ioslice_nc,gdcurv,w_end_d,w_buff,it_skip,t_end,0,wav->ncmp-1);
    }
    // snapshot
    io_snap_nc_put(iosnap, &iosnap_nc, gdcurv, md, wav, 
                   w_end_d, w_buff, nt_total, it, t_end, 1,1,1);

    // swap w_pre and w_end pointer, avoid copying
    w_cur_d = w_pre_d; w_pre_d = w_end_d; w_end_d = w_cur_d;
    f_cur_d = f_pre_d; f_pre_d = f_end_d; f_end_d = f_cur_d;

    if(bdrypml_d.is_enable_pml == 1)
    {
      for (int idim=0; idim<CONST_NDIM; idim++) {
        for (int iside=0; iside<2; iside++) {
          bdrypml_auxvar_t *auxvar_d = &(bdrypml_d.auxvar[idim][iside]);
          auxvar_d->cur = auxvar_d->pre;
          auxvar_d->pre = auxvar_d->end;
          auxvar_d->end = auxvar_d->cur;
        }
      }
    }
  } // time loop

  hipMemcpy(PG,PG_d,sizeof(float)*CONST_NDIM_5*gdcurv->ny*gdcurv->nx,hipMemcpyDeviceToHost);
  if (isfree == 1)
  {
    PG_slice_output(PG,gdcurv,output_dir,output_fname_part,topoid);
  }
  // io fault init_t0, peak_Vs at final time, use w_buff as buff
  io_fault_end_t_nc_put(&iofault_nc, gdcurv, fault_d, w_buff);

  // finish all time loop calculate, cudafree device pointer
  CUDACHECK(hipFree(PG_d));
  CUDACHECK(hipFree(Dis_accu_d));
  CUDACHECK(hipFree(neighid_d));
  dealloc_md_device(md_d);
  dealloc_metric_device(metric_d);
  dealloc_fd_device(fd_device_d);
  dealloc_fault_coef_device(fault_coef_d);
  dealloc_fault_device(fault_d);
  dealloc_fault_wav_device(fault_wav_d);
  dealloc_bdryfree_device(bdryfree_d);
  dealloc_bdrypml_device(bdrypml_d);
  dealloc_bdryexp_device(bdryexp_d);
  dealloc_wave_device(wav_d);

  // close nc
  io_fault_nc_close(&iofault_nc);
  io_slice_nc_close(&ioslice_nc);
  io_snap_nc_close(&iosnap_nc);

  return 0;
}

